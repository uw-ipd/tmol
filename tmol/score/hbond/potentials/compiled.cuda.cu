#include <tmol/score/common/simple_dispatch.cuda.impl.cuh>

#include "dispatch.impl.hh"

namespace tmol {
namespace score {
namespace hbond {
namespace potentials {
#define declare_dispatch(Real, Int)                                           \
  template struct HBondDispatch<AABBDispatch, tmol::Device::CUDA, Real, Int>; \
  template struct HBondDispatch<                                              \
      AABBTriuDispatch,                                                       \
      tmol::Device::CUDA,                                                     \
      Real,                                                                   \
      Int>;

declare_dispatch(float, int32_t);
declare_dispatch(double, int32_t);
declare_dispatch(float, int64_t);
declare_dispatch(double, int64_t);

}  // namespace potentials
}  // namespace hbond
}  // namespace score
}  // namespace tmol
