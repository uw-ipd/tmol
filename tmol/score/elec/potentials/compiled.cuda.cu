#include <tmol/score/common/forall_dispatch.cuda.impl.cuh>
#include <tmol/score/common/simple_dispatch.cuda.impl.cuh>

#include "dispatch.impl.hh"

namespace tmol {
namespace score {
namespace elec {
namespace potentials {

#define declare_dispatch(Real, Int) \
  template struct ElecDispatch<     \
      ForallDispatch,               \
      AABBDispatch,                 \
      tmol::Device::CUDA,           \
      Real,                         \
      Int>;                         \
  template struct ElecDispatch<     \
      ForallDispatch,               \
      AABBTriuDispatch,             \
      tmol::Device::CUDA,           \
      Real,                         \
      Int>;

declare_dispatch(float, int64_t);
declare_dispatch(double, int64_t);

}  // namespace potentials
}  // namespace elec
}  // namespace score
}  // namespace tmol
