#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <Eigen/Geometry>

#include <tmol/utility/tensor/TensorAccessor.h>
#include <tmol/utility/tensor/TensorPack.h>
#include <tmol/utility/tensor/TensorStruct.h>
#include <tmol/utility/tensor/TensorUtil.h>
#include <tmol/utility/nvtx.hh>

#include <tmol/score/common/accumulate.hh>
#include <tmol/score/common/coordinate_load.cuh>
#include <tmol/score/common/count_pair.hh>
#include <tmol/score/common/geom.hh>
#include <tmol/score/common/tuple.hh>

#include <tmol/score/ljlk/potentials/lj.hh>
#include <tmol/score/ljlk/potentials/rotamer_pair_energy_lj.hh>

#include <tmol/pack/sim_anneal/compiled/annealer.hh>

#include <chrono>

#include <tmol/score/common/forall_dispatch.cuda.impl.cuh>

#include <moderngpu/cta_load_balance.hxx>
#include <moderngpu/cta_reduce.hxx>
#include <moderngpu/cta_scan.hxx>
#include <moderngpu/cta_segreduce.hxx>
#include <moderngpu/cta_segscan.hxx>
#include <moderngpu/memory.hxx>
#include <moderngpu/search.hxx>
#include <moderngpu/transform.hxx>

// This file moves in more recent versions of Torch
#include <ATen/cuda/CUDAStream.h>

// #include <tmol/score/ljlk/potentials/rotamer_pair_energy_lj.impl.hh>

// The maximum number of inter-residue chemical bonds
#define MAX_N_CONN 4
#define TILE_SIZE 32

namespace tmol {
namespace score {
namespace ljlk {
namespace potentials {

static int already_printed = 0;

template <typename Real, int N>
using Vec = Eigen::Matrix<Real, N, 1>;

template <
    template <tmol::Device>
    class DeviceDispatch,
    tmol::Device D,
    typename Real,
    typename Int>
auto LJRPEDispatch<DeviceDispatch, D, Real, Int>::f(
    TView<Vec<Real, 3>, 3, D> context_coords,
    TView<Int, 2, D> context_block_type,
    TView<Vec<Real, 3>, 2, D> alternate_coords,
    TView<Vec<Int, 3>, 1, D>
        alternate_ids,  // 0 == context id; 1 == block id; 2 == block type

    // which system does a given context belong to
    TView<Int, 1, D> context_system_ids,

    // dims: n-systems x max-n-blocks x max-n-blocks
    // Quick lookup: given the inds of two blocks, ask: what is the minimum
    // number of chemical bonds that separate any pair of atoms in those blocks?
    // If this minimum is greater than the crossover, then no further logic for
    // deciding whether two atoms in those blocks should have their interaction
    // energies calculated: all should. intentionally small to (possibly) fit in
    // constant cache
    TView<Int, 3, D> system_min_bond_separation,

    // dims: n-systems x max-n-blocks x max-n-blocks x
    // max-n-interblock-connections x max-n-interblock-connections
    TView<Int, 5, D> system_inter_block_bondsep,

    // dims n-systems x max-n-blocks x max-n-neighbors
    // -1 as the sentinel
    TView<Int, 3, D> system_neighbor_list,

    //////////////////////
    // Chemical properties
    // how many atoms for a given block
    // Dimsize n_block_types
    TView<Int, 1, D> block_type_n_atoms,

    // what are the atom types for these atoms
    // Dimsize: n_block_types x max_n_atoms
    TView<Int, 2, D> block_type_atom_types,

    // how many inter-block chemical bonds are there
    // Dimsize: n_block_types
    TView<Int, 1, D> block_type_n_interblock_bonds,

    // what atoms form the inter-block chemical bonds
    // Dimsize: n_block_types x max_n_interblock_bonds
    TView<Int, 2, D> block_type_atoms_forming_chemical_bonds,

    // what is the path distance between pairs of atoms in the block
    // Dimsize: n_block_types x max_n_atoms x max_n_atoms
    TView<Int, 3, D> block_type_path_distance,
    //////////////////////

    // LJ parameters
    TView<LJTypeParams<Real>, 1, D> type_params,
    TView<LJGlobalParams<Real>, 1, D> global_params,
    TView<Real, 1, D> lj_lk_weights,
    TView<Real, 1, D> output) -> void {
  int const n_systems = system_min_bond_separation.size(0);
  int const n_contexts = context_coords.size(0);
  int64_t const n_alternate_blocks = alternate_coords.size(0);
  int const max_n_blocks = context_coords.size(1);
  int64_t const max_n_atoms = context_coords.size(2);
  int const n_block_types = block_type_n_atoms.size(0);
  int const max_n_interblock_bonds =
      block_type_atoms_forming_chemical_bonds.size(1);
  int64_t const max_n_neighbors = system_neighbor_list.size(2);
  int64_t const n_atom_types = type_params.size(0);

  assert(alternate_coords.size(1) == max_n_atoms);
  assert(alternate_ids.size(0) == n_alternate_blocks);
  assert(context_coords.size(0) == context_block_type.size(0));
  assert(context_system_ids.size(0) == n_contexts);

  assert(system_min_bond_separation.size(1) == max_n_blocks);
  assert(system_min_bond_separation.size(2) == max_n_blocks);

  assert(system_inter_block_bondsep.size(0) == n_systems);
  assert(system_inter_block_bondsep.size(1) == max_n_blocks);
  assert(system_inter_block_bondsep.size(2) == max_n_blocks);
  assert(system_inter_block_bondsep.size(3) == max_n_interblock_bonds);
  assert(system_inter_block_bondsep.size(4) == max_n_interblock_bonds);
  assert(system_neighbor_list.size(0) == n_systems);
  assert(system_neighbor_list.size(1) == max_n_blocks);

  assert(block_type_atom_types.size(0) == n_block_types);
  assert(block_type_atom_types.size(1) == max_n_atoms);
  assert(block_type_n_interblock_bonds.size(0) == n_block_types);
  assert(block_type_atoms_forming_chemical_bonds.size(0) == n_block_types);
  assert(block_type_path_distance.size(0) == n_block_types);
  assert(block_type_path_distance.size(1) == max_n_atoms);
  assert(block_type_path_distance.size(2) == max_n_atoms);

  assert(lj_lk_weights.size(0) == 2);

  // auto wcts = std::chrono::system_clock::now();
  // clock_t start_time = clock();

  // Allocate and zero the output tensors in a separate stream

  // auto output_t = TPack<Real, 1, D>::zeros({n_alternate_blocks});
  // auto output = output_t.view;
  // auto count_t = TPack<int, 1, D>::zeros({1});
  // auto count = count_t.view;
  //
  // // I'm not sure I want/need events for synchronization
  // auto event_t = TPack<int64_t, 1, D>::zeros({2});

  // return {output_t, event_t};

  static int call_count = 0;
  call_count += 1;

  int call_count_shadow = call_count;

  using namespace mgpu;
  typedef launch_box_t<
      arch_20_cta<64, 5>,
      arch_35_cta<64, 5>,
      arch_52_cta<64, 5>>
      launch_t;

  // between one alternate rotamer and its neighbors in the surrounding context
  auto score_inter_pairs = ([=] MGPU_DEVICE(
                                int tid,
                                int alt_start_atom,
                                int neighb_start_atom,
                                Real *alt_coords,
                                Real *neighb_coords,
                                LJTypeParams<Real> *alt_params,
                                LJTypeParams<Real> *neighb_params,
                                int const max_important_bond_separation,
                                int const alt_block_ind,
                                int const neighb_block_ind,
                                int const alt_block_type,
                                int const neighb_block_type,

                                int min_separation,
                                TensorAccessor<Int, 4, D> inter_block_bondsep,

                                int const alt_n_atoms,
                                int const neighb_n_atoms,
                                int const n_conn1,
                                int const n_conn2,
                                int const *path_dist1,
                                int const *path_dist2,
                                int const *conn_seps) {
    Real score_total = 0;
    Real coord1[3];
    Real coord2[3];

    int const alt_remain = min(TILE_SIZE, alt_n_atoms - alt_start_atom);
    int const neighb_remain =
        min(TILE_SIZE, neighb_n_atoms - neighb_start_atom);
    if (alt_remain < 0 || alt_remain > TILE_SIZE) {
      printf("error alt_remain %d\n", alt_remain);
    }
    if (neighb_remain < 0 || neighb_remain > TILE_SIZE) {
      printf("error neighb_remain %d\n", neighb_remain);
    }

    int const n_pairs = alt_remain * neighb_remain;

    LJGlobalParams<Real> global_params_local = global_params[0];
    Real lj_weight = lj_lk_weights[0];

    for (int i = tid; i < n_pairs; i += blockDim.x) {
      int const alt_atom_tile_ind = i / neighb_remain;
      int const neighb_atom_tile_ind = i % neighb_remain;

      if (alt_atom_tile_ind < 0 || alt_atom_tile_ind > TILE_SIZE) {
        printf("error alt_atom_tile_ind %d\n", alt_atom_tile_ind);
      }
      if (neighb_atom_tile_ind < 0 || neighb_atom_tile_ind > TILE_SIZE) {
        printf("error neighb_atom_tile_ind %d\n", neighb_atom_tile_ind);
      }

      int const alt_atom_ind = alt_atom_tile_ind + alt_start_atom;
      int const neighb_atom_ind = neighb_atom_tile_ind + neighb_start_atom;
      for (int j = 0; j < 3; ++j) {
        if (3 * alt_atom_tile_ind + j > 3 * TILE_SIZE) {
          printf(
              "errror 3 * alt_atom_tile_ind + j: %d\n",
              3 * alt_atom_tile_ind + j);
        }
        if (3 * neighb_atom_tile_ind + j > 3 * TILE_SIZE) {
          printf(
              "error 3 * neighb_atom_tile_ind + j: %d\n",
              3 * neighb_atom_tile_ind + j);
        }
        coord1[j] = alt_coords[3 * alt_atom_tile_ind + j];
        coord2[j] = neighb_coords[3 * neighb_atom_tile_ind + j];
      }

      // int const separation = 5;
      Real dist2 =
          ((coord1[0] - coord2[0]) * (coord1[0] - coord2[0])
           + (coord1[1] - coord2[1]) * (coord1[1] - coord2[1])
           + (coord1[2] - coord2[2]) * (coord1[2] - coord2[2]));
      if (dist2 > 36.0) {
        // DANGER -- maximum reach of LJ potential hard coded here in a second
        // place out of range!
        continue;
      }
      Real dist = std::sqrt(dist2);

      int separation = min_separation;
      if (separation <= max_important_bond_separation) {
        separation =
            common::count_pair::CountPair<D, Int>::inter_block_separation<
                TILE_SIZE>(
                max_important_bond_separation,
                alt_atom_tile_ind,
                neighb_atom_tile_ind,
                n_conn1,
                n_conn2,
                path_dist1,
                path_dist2,
                conn_seps);
      }

      // if (separation != separation2){
      //         printf("separation mismatch! %d %d %d %d %d\n", alt_atom_ind,
      // neighb_atom_ind, min_separation, separation, separation2);
      // }

      // TEMP short circuit the lennard-jones evaluation
      // Real lj = separation > 5 ? dist : 0;

      Real lj = lj_score<Real>::V(
          dist,
          separation,
          alt_params[alt_atom_tile_ind],
          neighb_params[neighb_atom_tile_ind],
          global_params_local);

      // Real lj = dist * separation *
      //   alt_params[alt_atom_tile_ind].lj_radius *
      //   neighb_params[neighb_atom_tile_ind].lj_radius *
      //   global_params_local.lj_hbond_dis;

      lj *= lj_weight;

      // if ( lj != 0 ) {
      //   printf("cuda %d %d %6.3f %6.3f %6.3f vs %6.3f %6.3f %6.3f e=
      //   %8.4f\n",
      //     alt_atom_ind, neighb_atom_ind,
      //     coord1[0], coord1[1], coord1[2],
      //     coord2[0], coord2[1], coord2[2],
      //     lj
      //   );
      // }

      score_total += lj;
    }
    return score_total;
  });

  // between one atoms within an alternate rotamer
  auto score_intra_pairs = ([=] MGPU_DEVICE(
                                int tid,
                                int start_atom1,
                                int start_atom2,
                                Real *coords1,
                                Real *coords2,
                                LJTypeParams<Real> *params1,
                                LJTypeParams<Real> *params2,
                                int const max_important_bond_separation,
                                int const block_type,
                                int const n_atoms) {
    Real score_total = 0;
    Real coord1[3];
    Real coord2[3];

    int const remain1 = min(TILE_SIZE, n_atoms - start_atom1);
    int const remain2 = min(TILE_SIZE, n_atoms - start_atom2);

    int const n_pairs = remain1 * remain2;

    LJGlobalParams<Real> global_params_local = global_params[0];
    Real lj_weight = lj_lk_weights[0];

    for (int i = tid; i < n_pairs; i += blockDim.x) {
      int const atom_ind_1_local = i / remain2;
      int const atom_ind_2_local = i % remain2;
      int const atom_ind_1 = atom_ind_1_local + start_atom1;
      int const atom_ind_2 = atom_ind_2_local + start_atom2;
      if (atom_ind_1 >= atom_ind_2) {
        continue;
      }

      for (int j = 0; j < 3; ++j) {
        coord1[j] = coords1[3 * atom_ind_1_local + j];
        coord2[j] = coords2[3 * atom_ind_2_local + j];
      }
      // int const atom_1_type = atom_type1[atom_ind_1_local];
      // int const atom_2_type = atom_type2[atom_ind_2_local];

      int const separation =
          block_type_path_distance[block_type][atom_ind_1][atom_ind_2];

      Real const dist = sqrt(
          (coord1[0] - coord2[0]) * (coord1[0] - coord2[0])
          + (coord1[1] - coord2[1]) * (coord1[1] - coord2[1])
          + (coord1[2] - coord2[2]) * (coord1[2] - coord2[2]));

      Real lj = lj_score<Real>::V(
          dist,
          separation,
          params1[atom_ind_1_local],
          params2[atom_ind_2_local],
          global_params_local);
      lj *= lj_lk_weights[0];
      score_total += lj;
    }
    return score_total;
  });

  auto eval_energies = ([=] MGPU_DEVICE(int tid, int cta) {
    typedef typename launch_t::sm_ptx params_t;
    enum {
      nt = params_t::nt,
      vt = params_t::vt,
      vt0 = params_t::vt0,
      nv = nt * vt
    };
    typedef mgpu::cta_reduce_t<nt, Real> reduce_t;

    // struct struct_part1 {
    //   Real coords_alt1[TILE_SIZE * 3];  // 786 bytes for coords
    //   Real coords_alt2[TILE_SIZE * 3];
    //   LJTypeParams<Real> params_alt1[TILE_SIZE];  // 1536 bytes for params
    //   LJTypeParams<Real> params_alt2[TILE_SIZE];
    //   Int min_separation;  // 8 bytes for two integers
    //   Int n_conn_alt;
    //   Int conn_ats_alt1[MAX_N_CONN];  // 32 bytes for conn ats
    //   Int conn_ats_alt2[MAX_N_CONN];
    //   Int path_dist_alt1[MAX_N_CONN * TILE_SIZE];  // 1024 for path dists
    //   Int path_dist_alt2[MAX_N_CONN * TILE_SIZE];
    // };

    __shared__ struct shared_mem_struct {
      Real coords_alt1[TILE_SIZE * 3];  // 786 bytes for coords
      Real coords_alt2[TILE_SIZE * 3];
      LJTypeParams<Real> params_alt1[TILE_SIZE];  // 1536 bytes for params
      LJTypeParams<Real> params_alt2[TILE_SIZE];
      Int min_separation;  // 8 bytes for two integers
      Int n_conn_alt;
      Int conn_ats_alt1[MAX_N_CONN];  // TILE_SIZE bytes for conn ats
      Int conn_ats_alt2[MAX_N_CONN];
      Int path_dist_alt1[MAX_N_CONN * TILE_SIZE];  // 1024 for path dists
      Int path_dist_alt2[MAX_N_CONN * TILE_SIZE];

      union union_pt2_red {
        struct struct_part2 {
          Real coords_other[TILE_SIZE * 3];             // 384 bytes for coords
          Int n_conn_other;                             // 4 bytes for an int
          LJTypeParams<Real> params_other[TILE_SIZE];   // 768 bytes for params
          Int conn_ats_other[MAX_N_CONN];               // 16 bytes
          Int path_dist_other[MAX_N_CONN * TILE_SIZE];  // 512 bypes
          Int conn_seps[MAX_N_CONN * MAX_N_CONN];  // 64 bytes for conn/conn
        } vals;
        typename reduce_t::storage_t reduce;

      } union_vals;
      // bool bad; // TEMP!

    } shared;

    // if (false) {
    // // if (cta == 0 and tid == 0) {
    //   printf("sizeof shared_mem_struct %lu, reduce size %lu, pt1 %lu, pt2
    //   %lu, union %lu\n",
    //         sizeof(shared_mem_struct),
    //         sizeof(reduce_t::storage_t),
    //         sizeof(struct_part1),
    //         sizeof(shared_mem_struct::union_pt2_red::struct_part2),
    //         sizeof(shared_mem_struct::union_pt2_red)
    //   );
    // }

    Real *coords_alt1 = shared.coords_alt1;
    Real *coords_alt2 = shared.coords_alt2;
    Real *coords_other = shared.union_vals.vals.coords_other;
    LJTypeParams<Real> *params_alt1 = shared.params_alt1;
    LJTypeParams<Real> *params_alt2 = shared.params_alt2;
    LJTypeParams<Real> *params_other = shared.union_vals.vals.params_other;
    if (tid == 0) {
      // shared.bad = false;
    }

    Int last_alt_ind = -1;
    bool count_pair_data_loaded = false;

    for (int iteration = 0; iteration < vt; ++iteration) {
      Real totalE1 = 0;
      Real totalE2 = 0;

      int alt_ind = (vt * cta + iteration) / max_n_neighbors;

      if (alt_ind >= n_alternate_blocks / 2) {
        break;
      }
      bool const new_alt = alt_ind != last_alt_ind;
      // last_alt_ind = alt_ind;
      if (new_alt) {
        count_pair_data_loaded = false;
      }

      int neighb_ind = (vt * cta + iteration) % max_n_neighbors;

      int const max_important_bond_separation = 4;
      int const alt_context = alternate_ids[2 * alt_ind][0];
      if (alt_context == -1) {
        continue;
      }

      if (iteration == 0 && cta == 0 && tid == 0
          && call_count_shadow % 100 == 1) {
        printf("abi %d\n", alternate_ids[2 * alt_ind][1]);
      }

      int const alt_block_ind = alternate_ids[2 * alt_ind][1];
      int const alt_block_type1 = alternate_ids[2 * alt_ind][2];
      int const alt_block_type2 = alternate_ids[2 * alt_ind + 1][2];
      // if (tid == 0) {
      //         printf("alt block type: %d ind, %d type1, %d ind type2\n",
      //         alt_ind, alt_block_type1, alt_block_type2);
      // }
      if (alt_context >= n_contexts || alt_context < 0) {
        printf("Error alt_context %d\n", alt_context);
      }
      int const system = context_system_ids[alt_context];
      if (system < 0 || system >= n_systems) {
        printf("Error system %d\n", system);
      }
      if (alt_block_type1 >= n_block_types || alt_block_type1 < 0) {
        printf("Error alt_block_type1 %d\n", alt_block_type1);
      }
      if (alt_block_type2 >= n_block_types || alt_block_type2 < 0) {
        printf("Error alt_block_type2 %d\n", alt_block_type2);
      }
      int const alt_n_atoms1 = block_type_n_atoms[alt_block_type1];
      int const alt_n_atoms2 = block_type_n_atoms[alt_block_type2];
      if (alt_n_atoms1 < 0 || alt_n_atoms1 > 100) {
        printf("error alt_n_atoms1 %d\n", alt_n_atoms1);
      }
      if (alt_n_atoms2 < 0 || alt_n_atoms2 > 100) {
        printf("error alt_n_atoms2 %d\n", alt_n_atoms2);
      }

      int const neighb_block_ind =
          system_neighbor_list[system][alt_block_ind][neighb_ind];
      if (neighb_block_ind == -1) {
        continue;
      }

      int n_conn_other(-1);

      if (alt_block_ind != neighb_block_ind) {
        if (alt_block_ind >= max_n_blocks || alt_block_ind < 0) {
          printf("Error alt_block_ind %d\n", alt_block_ind);
        }
        if (neighb_block_ind >= max_n_blocks || neighb_block_ind < 0) {
          printf("Error neighb_block_ind %d\n", neighb_block_ind);
        }
        // inter-residue energy evaluation

        int const neighb_block_type =
            context_block_type[alt_context][neighb_block_ind];
        if (neighb_block_type >= n_block_types || neighb_block_type < 0) {
          printf("Error neighb_block_type %d\n", neighb_block_type);
        }
        int const neighb_n_atoms = block_type_n_atoms[neighb_block_type];

        int const n_conn_alt_x = block_type_n_interblock_bonds[alt_block_type1];
        int const n_conn_other_x =
            block_type_n_interblock_bonds[neighb_block_type];
        int const min_sep_x =
            system_min_bond_separation[system][alt_block_ind][neighb_block_ind];
        __syncthreads();
        if (tid == 0) {
          // printf("min_sep %2d\n", min_sep);
          int const n_conn_alt_s =
              block_type_n_interblock_bonds[alt_block_type1];
          int const n_conn_other_s =
              block_type_n_interblock_bonds[neighb_block_type];
          int const min_sep_s =
              system_min_bond_separation[system][alt_block_ind]
                                        [neighb_block_ind];
          if (n_conn_alt_s >= MAX_N_CONN) {
            printf("Error n_conn_alt %d\n", n_conn_alt_s);
          }
          if (n_conn_other_s >= MAX_N_CONN) {
            printf("Error n_conn_other_s %d\n", n_conn_other_s);
          }
          shared.min_separation = min_sep_s;
          shared.n_conn_alt = n_conn_alt_s;
          shared.union_vals.vals.n_conn_other = n_conn_other_s;
        }
        __syncthreads();

        int const min_sep = shared.min_separation;

        bool const count_pair_striking_dist =
            min_sep <= max_important_bond_separation;

        int const n_conn_alt = shared.n_conn_alt;
        n_conn_other = shared.union_vals.vals.n_conn_other;

        if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
          printf("n_conn_other discrepancy 1\n");
        }

        if (n_conn_alt >= MAX_N_CONN) {
          printf(
              "Error n_conn_alt %d %d %d %d %d\n",
              n_conn_alt,
              block_type_n_interblock_bonds[alt_block_type1],
              tid,
              alt_block_ind,
              alt_block_type1);
        }
        if (n_conn_other >= MAX_N_CONN) {
          int blah = shared.union_vals.vals.n_conn_other;
          if (tid % 32 == 0) {
            // shared.bad = true;
            printf(
                "Error n_conn_other %d %d %d %d %d %d %d %d\n",
                n_conn_other,
                block_type_n_interblock_bonds[neighb_block_type],
                call_count_shadow,
                tid,
                neighb_block_ind,
                neighb_block_type,
                iteration,
                blah);
            // printf("reduce data: ");
            // for (int ii = 0; ii < max(nt, 2 * min(nt, 32)); ++ii) {
            //   printf(" %d %d %f", ii, *(reinterpret_cast<int *>
            //   (&shared.union_vals.reduce.data[ii])), (float)
            //   shared.union_vals.reduce.data[ii]);
            // }
            // printf("\n");
            // printf("addresses:");
            // printf("coords alt1 %p\n", &shared.coords_alt1);
            // printf("coords alt2 %p\n", &shared.coords_alt2);
            // printf("params alt1 %p\n", &shared.params_alt1);
            // printf("params alt2 %p\n", &shared.params_alt2);
            // printf("min sep %p\n", &shared.min_separation);
            // printf("n_conn_alt %p\n", &shared.n_conn_alt);
            // printf("conn ats alt1 %p\n", &shared.conn_ats_alt1);
            // printf("conn ats alt2 %p\n", &shared.conn_ats_alt2);
            // printf("path_dist_alt1 %p\n", &shared.path_dist_alt1);
            // printf("path_dist_alt2 %p\n", &shared.path_dist_alt2);
            // printf("shared.union_vals.vals.coords_other %p\n", &
            // shared.union_vals.vals.coords_other);
            // printf("shared.union_vals.vals.n_conn_other %p\n", &
            // shared.union_vals.vals.n_conn_other);
            // printf("shared.union_vals.vals.params_other %p\n", &
            // shared.union_vals.vals.params_other);
            // printf("shared.union_vals.vals.conn_ats_other %p\n", &
            // shared.union_vals.vals.conn_ats_other);
            // printf("shared.union_vals.vals.path_dist_other %p\n", &
            // shared.union_vals.vals.path_dist_other);
            // printf("shared.union_vals.vals.conn_ats_other %p\n", &
            // shared.union_vals.vals.conn_seps);
            // printf("shared.union_vals.reduce %p\n", &
            // shared.union_vals.reduce);
          }

          // FIX IT BEFORE CONTINUING ON
          n_conn_other = block_type_n_interblock_bonds[neighb_block_type];
          shared.union_vals.vals.n_conn_other = 5;
        }
        __syncthreads();

        if (count_pair_striking_dist && tid < n_conn_alt) {
          shared.conn_ats_alt1[tid] =
              block_type_atoms_forming_chemical_bonds[alt_block_type1][tid];
          shared.conn_ats_alt2[tid] =
              block_type_atoms_forming_chemical_bonds[alt_block_type2][tid];
        }
        if (count_pair_striking_dist && tid < n_conn_other) {
          shared.union_vals.vals.conn_ats_other[tid] =
              block_type_atoms_forming_chemical_bonds[neighb_block_type][tid];
        }
        if (count_pair_striking_dist && tid < n_conn_alt * n_conn_other) {
          if (tid >= 16) {
            printf("conn alt * conn other error\n");
          }
          int conn1 = tid / n_conn_other;
          int conn2 = tid % n_conn_other;
          shared.union_vals.vals.conn_seps[tid] =
              system_inter_block_bondsep[system][alt_block_ind]
                                        [neighb_block_ind][conn1][conn2];
        }
        if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
          printf("n_conn_other discrepancy 2\n");
        }
        __syncthreads();

        // Tile the sets of TILE_SIZE atoms
        int const alt_n_iterations =
            (max(alt_n_atoms1, alt_n_atoms2) - 4 - 1) / TILE_SIZE + 1;
        int const neighb_n_iterations =
            (neighb_n_atoms - 4 - 1) / TILE_SIZE + 1;

        for (int i = 0; i < alt_n_iterations; ++i) {
          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 3\n");
          }
          if (i != 0) {
            // make sure all threads have completed their work
            // from the previous iteration before we overwrite
            // the contents of shared memory
            __syncthreads();
          }

          // Let's load coordinates and Lennard-Jones parameters for
          // TILE_SIZE atoms into shared memory
          int const i_n_atoms_to_load1 = max(
              0, min(Int(TILE_SIZE), Int((alt_n_atoms1 - TILE_SIZE * i - 4))));

          int const i_n_atoms_to_load2 = max(
              0, min(Int(TILE_SIZE), Int((alt_n_atoms2 - TILE_SIZE * i - 4))));

          // continue; // BAD?!?!
          if (new_alt || alt_n_atoms1 > TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(
                    &alternate_coords[2 * alt_ind][4 + i * TILE_SIZE]),
                tid,
                i_n_atoms_to_load1 * 3,
                coords_alt1,
                false);
          }

          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 4\n");
          }
          if (new_alt || alt_n_atoms2 > TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(
                    &alternate_coords[2 * alt_ind + 1][4 + i * TILE_SIZE]),
                tid,
                i_n_atoms_to_load2 * 3,
                coords_alt2,
                false);
          }

          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 5\n");
          }
          // continue; //  BAD?!!

          if ((new_alt || alt_n_atoms1 > TILE_SIZE) && tid < TILE_SIZE) {
            // coalesced read of atom coordinate data
            // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
            //     alternate_coords[2 * alt_ind], i * TILE_SIZE + 4,
            //     coords_alt1, tid);

            // load the Lennard-Jones parameters for these TILE_SIZE atoms
            if (tid < i_n_atoms_to_load1) {
              int const atid = TILE_SIZE * i + tid + 4;
              if (atid >= max_n_atoms || atid < 0) {
                printf("error atid %d\n", atid);
              }
              int const attype = block_type_atom_types[alt_block_type1][atid];
              if (attype >= 0) {
                if (attype >= n_atom_types) {
                  printf("error attype %d\n", attype);
                }
                params_alt1[tid] = type_params[attype];
              }
            }
          }

          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 6\n");
          }
          if ((new_alt || alt_n_atoms1 > TILE_SIZE || !count_pair_data_loaded)
              && tid < i_n_atoms_to_load1) {
            int const atid = TILE_SIZE * i + tid + 4;
            if (atid >= max_n_atoms || atid < 0) {
              printf("error atid %d\n", atid);
            }
            if (count_pair_striking_dist && !count_pair_data_loaded) {
              for (int j = 0; j < n_conn_alt; ++j) {
                if (shared.conn_ats_alt1[j] >= max_n_atoms
                    || shared.conn_ats_alt1[j] < 0) {
                  printf(
                      "error conn_ats_alt1[j] %d\n", shared.conn_ats_alt1[j]);
                }
                int ij_path_dist =
                    block_type_path_distance[alt_block_type1]
                                            [shared.conn_ats_alt1[j]][atid];
                if (j * TILE_SIZE + tid > MAX_N_CONN * TILE_SIZE) {
                  printf("error storing path dists: %d\n", j * TILE_SIZE + tid);
                }
                shared.path_dist_alt1[j * TILE_SIZE + tid] = ij_path_dist;
              }
            }
          }

          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 7\n");
          }
          // continue; // GOOD

          if ((new_alt || alt_n_atoms2 > TILE_SIZE)
              && tid < i_n_atoms_to_load2) {
            // load the Lennard-Jones parameters for these TILE_SIZE atoms

            int const atid = TILE_SIZE * i + tid + 4;
            if (atid >= max_n_atoms || atid < 0) {
              printf("error atid %d\n", atid);
            }
            int const attype = block_type_atom_types[alt_block_type2][atid];

            // printf("alt_block_ind %d, atid %d, attype %d, max_n_params %d\n",
            // alt_block_ind, atid, attype, int(type_params.size(0)));
            if (attype >= 0) {
              if (attype >= n_atom_types) {
                printf("error attype %d\n", attype);
              }
              params_alt2[tid] = type_params[attype];
            }
          }

          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 8\n");
          }
          // continue; // BAD??!!

          if ((new_alt || alt_n_atoms2 > TILE_SIZE || !count_pair_data_loaded)
              && tid < i_n_atoms_to_load2) {
            int const atid = TILE_SIZE * i + tid + 4;
            if (atid >= max_n_atoms || atid < 0) {
              printf("error atid %d\n", atid);
            }
            if (count_pair_striking_dist && !count_pair_data_loaded) {
              for (int j = 0; j < n_conn_alt; ++j) {
                if (shared.conn_ats_alt2[j] >= max_n_atoms
                    || shared.conn_ats_alt2[j] < 0) {
                  printf(
                      "error conn_ats_alt2[j] %d\n", shared.conn_ats_alt2[j]);
                }
                int ij_path_dist =
                    block_type_path_distance[alt_block_type2]
                                            [shared.conn_ats_alt2[j]][atid];
                if (j * TILE_SIZE + tid > MAX_N_CONN * TILE_SIZE) {
                  printf("error storing path dists: %d\n", j * TILE_SIZE + tid);
                }
                shared.path_dist_alt2[j * TILE_SIZE + tid] = ij_path_dist;
              }
            }
          }
          if (tid < 32 && shared.union_vals.vals.n_conn_other != n_conn_other) {
            printf("n_conn_other discrepancy 9\n");
          }
          if (count_pair_striking_dist) {
            count_pair_data_loaded = true;
          }
          // continue; // BAD

          for (int j = 0; j < neighb_n_iterations; ++j) {
            if (j != 0) {
              // make sure that all threads have finished energy
              // calculations from the previous iteration
              __syncthreads();
            }
            int j_n_atoms_to_load =
                min(Int(TILE_SIZE), Int((neighb_n_atoms - TILE_SIZE * j - 4)));
            // if (j_n_atoms_to_load >= TILE_SIZE) {
            // if (tid == 0 && call_counts_shadow % 100 == 1) {
            //   printf("j_n_atoms_to_load %d\n", j_n_atoms_to_load);
            // }
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(
                    &context_coords[alt_context][neighb_block_ind]
                                   [4 + j * TILE_SIZE]),
                tid,
                j_n_atoms_to_load * 3,
                coords_other,
                false);
            if (tid < 32
                && shared.union_vals.vals.n_conn_other != n_conn_other) {
              printf("n_conn_other discrepancy 10\n");
            }
            // if ( tid < j_n_atoms_to_load ) {
            //   Vec<Real, 3> coord =
            //   context_coords[alt_context][neighb_block_ind][tid + 4 + j *
            //   TILE_SIZE]; coords_other[3 * tid + 0] = coord[ 0 ];
            //   coords_other[3 * tid + 1] = coord[ 1 ];
            //   coords_other[3 * tid + 2] = coord[ 2 ];
            // }

            if (tid < TILE_SIZE) {
              // Coalesced read of atom coordinate data
              // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
              //     context_coords[alt_context][neighb_block_ind],
              //     j * TILE_SIZE + 4,
              //     coords_other,
              //     tid);

              // load the Lennard-Jones parameters for these TILE_SIZE atoms
              if (tid < j_n_atoms_to_load) {
                int const atid = TILE_SIZE * j + 4 + tid;
                if (atid >= max_n_atoms || atid < 0) {
                  printf("error atid %d\n", atid);
                }
                int const attype =
                    block_type_atom_types[neighb_block_type][atid];
                if (attype >= 0) {
                  if (attype >= n_atom_types) {
                    printf("error attype %d\n", attype);
                  }
                  params_other[tid] = type_params[attype];
                }
                if (tid < 32
                    && shared.union_vals.vals.n_conn_other != n_conn_other) {
                  printf("n_conn_other discrepancy 11\n");
                }
                if (count_pair_striking_dist) {
                  for (int k = 0; k < n_conn_other; ++k) {
                    if (shared.union_vals.vals.conn_ats_other[k] >= max_n_atoms
                        || shared.union_vals.vals.conn_ats_other[k] < 0) {
                      printf(
                          "error conn_ats_other[j] %d\n",
                          shared.union_vals.vals.conn_ats_other[k]);
                    }
                    int jk_path_dist =
                        block_type_path_distance[neighb_block_type]
                                                [shared.union_vals.vals
                                                     .conn_ats_other[k]][atid];
                    if (k * TILE_SIZE + tid > MAX_N_CONN * TILE_SIZE) {
                      printf(
                          "error storing path dists: %d\n",
                          k * TILE_SIZE + tid);
                    }
                    shared.union_vals.vals
                        .path_dist_other[k * TILE_SIZE + tid] = jk_path_dist;
                  }
                }
              }
            }

            // make sure shared-memory loading has completed before we proceed
            // into energy calculations
            __syncthreads();
            if (tid < 32
                && shared.union_vals.vals.n_conn_other != n_conn_other) {
              printf("n_conn_other discrepancy 12\n");
            }

            // Now we will calculate the TILE_SIZExTILE_SIZE atom pair energies
            totalE1 = score_inter_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt1,
                coords_other,
                params_alt1,
                params_other,
                max_important_bond_separation,
                alt_block_ind,
                neighb_block_ind,
                alt_block_type1,
                neighb_block_type,
                min_sep,
                system_inter_block_bondsep[system],
                alt_n_atoms1,
                neighb_n_atoms,
                n_conn_alt,
                n_conn_other,
                shared.path_dist_alt1,
                shared.union_vals.vals.path_dist_other,
                shared.union_vals.vals.conn_seps);

            if (tid < 32
                && shared.union_vals.vals.n_conn_other != n_conn_other) {
              printf("n_conn_other discrepancy 13\n");
            }
            totalE2 = score_inter_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt2,
                coords_other,
                params_alt2,
                params_other,
                max_important_bond_separation,
                alt_block_ind,
                neighb_block_ind,
                alt_block_type2,
                neighb_block_type,
                min_sep,
                system_inter_block_bondsep[system],
                alt_n_atoms2,
                neighb_n_atoms,
                n_conn_alt,
                n_conn_other,
                shared.path_dist_alt2,
                shared.union_vals.vals.path_dist_other,
                shared.union_vals.vals.conn_seps);
            if (tid < 32
                && shared.union_vals.vals.n_conn_other != n_conn_other) {
              printf("n_conn_other discrepancy 14\n");
            }
          }  // for j
        }    // for i
      } else {
        // alt_block_ind == neighb_block_ind
        // continue; // TEMP! Skip intra-residue to debug inter-residue

        // int const alt_n_atoms = block_type_n_atoms[alt_block_type];

        int const n_iterations =
            (max(alt_n_atoms1, alt_n_atoms2) - 4 - 1) / TILE_SIZE + 1;

        for (int i = 0; i < n_iterations; ++i) {
          if (i != 0) {
            // make sure the calculations for the previous iteration
            // have completed before we overwrite the contents of
            // shared memory
            __syncthreads();
          }
          int const i_n_atoms_to_load1 =
              min(Int(TILE_SIZE), Int((alt_n_atoms1 - TILE_SIZE * i - 4)));

          int const i_n_atoms_to_load2 =
              min(Int(TILE_SIZE), Int((alt_n_atoms2 - TILE_SIZE * i - 4)));

          if ((new_alt || alt_n_atoms1 > TILE_SIZE) && tid < TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind][4]),
                tid,
                i_n_atoms_to_load1 * 3,
                coords_alt1,
                false);

            // load Lennard-Jones parameters for the TILE_SIZE atoms into shared
            // memory
            if (i * TILE_SIZE + 4 + tid < max_n_atoms) {
              int const atind = i * TILE_SIZE + tid + 4;
              int const attype = block_type_atom_types[alt_block_type1][atind];
              if (attype >= 0) {
                params_alt1[tid] = type_params[attype];
              }
            }
          }
          if ((new_alt || alt_n_atoms2 > TILE_SIZE) && tid < TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind + 1][4]),
                tid,
                i_n_atoms_to_load2 * 3,
                coords_alt2,
                false);
            // coalesced reads of coordinate data
            // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
            //     alternate_coords[2 * alt_ind + 1],
            //     i * TILE_SIZE + 4,
            //     coords_alt2,
            //     tid);

            // load Lennard-Jones parameters for the TILE_SIZE atoms into shared
            // memory
            if (i * TILE_SIZE + 4 + tid < max_n_atoms) {
              int const atind = i * TILE_SIZE + tid + 4;
              int const attype = block_type_atom_types[alt_block_type2][atind];
              if (attype >= 0) {
                params_alt2[tid] = type_params[attype];
              }
            }
          }

          // process residue 1
          for (int j = i; j < n_iterations; ++j) {
            if (j != i) {
              // make sure calculations from the previous iteration have
              // completed before we overwrite the contents of shared
              // memory
              __syncthreads();
            }

            if (j != i && tid < TILE_SIZE) {
              mgpu::mem_to_shared<TILE_SIZE, 3>(
                  reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind][4]),
                  tid,
                  i_n_atoms_to_load1 * 3,
                  coords_other,
                  false);
              // coalesced read of coordinate data
              // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
              //     alternate_coords[2 * alt_ind], j * TILE_SIZE + 4,
              //     coords_other, tid);
              if (j * TILE_SIZE + tid < max_n_atoms) {
                int const atind = j * TILE_SIZE + 4 + tid;
                int const attype =
                    block_type_atom_types[alt_block_type1][atind];
                if (attype >= 0) {
                  params_other[tid] = type_params[attype];
                }
              }
            }
            __syncthreads();
            totalE1 = score_intra_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt1,
                (i == j ? coords_alt1 : coords_other),
                params_alt1,
                (i == j ? params_alt1 : params_other),
                max_important_bond_separation,
                alt_block_type1,
                alt_n_atoms1);
          }  // for j

          // Process residue 2
          for (int j = i; j < n_iterations; ++j) {
            if (j != i) {
              // make sure calculations from the previous iteration have
              // completed before we overwrite the contents of shared
              // memory
              __syncthreads();
            }

            if (j != i && tid < TILE_SIZE) {
              mgpu::mem_to_shared<TILE_SIZE, 3>(
                  reinterpret_cast<Real *>(
                      &alternate_coords[2 * alt_ind + 1][4]),
                  tid,
                  i_n_atoms_to_load2 * 3,
                  coords_other,
                  false);
              if (j * TILE_SIZE + tid < max_n_atoms) {
                int const atind = j * TILE_SIZE + 4 + tid;
                int const attype =
                    block_type_atom_types[alt_block_type2][atind];
                if (attype >= 0) {
                  params_other[tid] = type_params[attype];
                }
              }
            }
            __syncthreads();
            totalE2 = score_intra_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt2,
                (i == j ? coords_alt2 : coords_other),
                params_alt2,
                (i == j ? params_alt2 : params_other),
                max_important_bond_separation,
                alt_block_type2,
                alt_n_atoms2);
          }  // for j
        }    // for i
      }      // else

      if (alt_block_ind != neighb_block_ind && tid < 32
          && shared.union_vals.vals.n_conn_other != n_conn_other) {
        printf("n_conn_other discrepancy 15\n");
      }
      __syncthreads();

      Real const cta_totalE1 = reduce_t().reduce(
          tid, totalE1, shared.union_vals.reduce, nt, mgpu::plus_t<Real>());

      __syncthreads();

      Real const cta_totalE2 = reduce_t().reduce(
          tid, totalE2, shared.union_vals.reduce, nt, mgpu::plus_t<Real>());

      if (tid == 0) {
        // printf("%d %d %f; %d %d %f\n", 2 * alt_ind, neighb_ind, cta_totalE1,
        // 2 * alt_ind + 1, neighb_ind, cta_totalE2);
        atomicAdd(&output[2 * alt_ind], cta_totalE1);
        atomicAdd(&output[2 * alt_ind + 1], cta_totalE2);
      }

      if (alt_block_ind != neighb_block_ind && tid < 32
          && shared.union_vals.vals.n_conn_other != n_conn_other) {
        printf("n_conn_other discrepancy 16\n");
      }
      __syncthreads();
      shared.union_vals.reduce.data[tid] = 11.1 * tid;
      if (shared.union_vals.vals.n_conn_other > MAX_N_CONN) {
        if (tid == 32 || tid == 0) {
          printf(
              "Error x n_conn_other %d %d\n",
              shared.union_vals.vals.n_conn_other,
              tid);
        }
      }
      if (alt_block_ind != neighb_block_ind && tid < 32
          && shared.union_vals.vals.n_conn_other != n_conn_other) {
        printf("n_conn_other discrepancy 17\n");
      }
    }  // for iteration
  });

  at::cuda::CUDAStream wrapped_stream = at::cuda::getStreamFromPool();
  setCurrentCUDAStream(wrapped_stream);
  mgpu::standard_context_t context(wrapped_stream.stream());

  // mgpu::standard_context_t context;

  int const n_ctas =
      (n_alternate_blocks * max_n_neighbors / 2 - 1) / launch_t::sm_ptx::vt + 1;
  if (already_printed == 0) {
    std::cout << "n_ctas: " << n_ctas << " n_alternate_blocks "
              << n_alternate_blocks << " max_n_neighbors " << max_n_neighbors
              << std::endl;
    already_printed = 1;
  }
  mgpu::cta_launch<launch_t>(eval_energies, n_ctas, context);

  at::cuda::setCurrentCUDAStream(at::cuda::getDefaultCUDAStream());

#ifdef __HIPCC__
  // float first;
  // hipMemcpy(&first, &output[0], sizeof(float), hipMemcpyDeviceToHost);
  //
  // clock_t stop_time = clock();
  // std::chrono::duration<double> wctduration =
  // (std::chrono::system_clock::now() - wcts);
  //
  // std::cout << n_systems << " " << n_contexts << " " <<n_alternate_blocks <<
  // " "; std::cout << n_alternate_blocks * max_n_neighbors * max_n_atoms *
  // max_n_atoms << " "; std::cout << "runtime? " << ((double)stop_time -
  // start_time) / CLOCKS_PER_SEC
  //           << " wall time: " << wctduration.count() << " " << first
  //           << std::endl;
#endif
  // return {output_t, event_t};
}

template <
    template <tmol::Device>
    class DeviceDispatch,
    tmol::Device D,
    typename Real,
    typename Int>
class LJRPECudaCalc : public pack::sim_anneal::compiled::RPECalc {
 public:
  LJRPECudaCalc(
      TView<Vec<Real, 3>, 3, D> context_coords,
      TView<Int, 2, D> context_block_type,
      TView<Vec<Real, 3>, 2, D> alternate_coords,
      TView<Vec<Int, 3>, 1, D>
          alternate_ids,  // 0 == context id; 1 == block id; 2 == block type

      // which system does a given context belong to
      TView<Int, 1, D> context_system_ids,

      // dims: n-systems x max-n-blocks x max-n-blocks
      // Quick lookup: given the inds of two blocks, ask: what is the minimum
      // number of chemical bonds that separate any pair of atoms in those
      // blocks? If this minimum is greater than the crossover, then no further
      // logic for deciding whether two atoms in those blocks should have their
      // interaction energies calculated: all should. intentionally small to
      // (possibly) fit in constant cache
      TView<Int, 3, D> system_min_bond_separation,

      // dims: n-systems x max-n-blocks x max-n-blocks x
      // max-n-interblock-connections x max-n-interblock-connections
      TView<Int, 5, D> system_inter_block_bondsep,

      // dims n-systems x max-n-blocks x max-n-neighbors
      // -1 as the sentinel
      TView<Int, 3, D> system_neighbor_list,

      //////////////////////
      // Chemical properties
      // how many atoms for a given block
      // Dimsize n_block_types
      TView<Int, 1, D> block_type_n_atoms,

      // what are the atom types for these atoms
      // Dimsize: n_block_types x max_n_atoms
      TView<Int, 2, D> block_type_atom_types,

      // how many inter-block chemical bonds are there
      // Dimsize: n_block_types
      TView<Int, 1, D> block_type_n_interblock_bonds,

      // what atoms form the inter-block chemical bonds
      // Dimsize: n_block_types x max_n_interblock_bonds
      TView<Int, 2, D> block_type_atoms_forming_chemical_bonds,

      // what is the path distance between pairs of atoms in the block
      // Dimsize: n_block_types x max_n_atoms x max_n_atoms
      TView<Int, 3, D> block_type_path_distance,
      //////////////////////

      // LJ parameters
      TView<LJTypeParams<Real>, 1, D> type_params,
      TView<LJGlobalParams<Real>, 1, D> global_params,
      TView<Real, 1, D> lj_lk_weights,
      TView<Real, 1, D> output)
      : context_coords_(context_coords),
        context_block_type_(context_block_type),
        alternate_coords_(alternate_coords),
        alternate_ids_(alternate_ids),
        context_system_ids_(context_system_ids),
        system_min_bond_separation_(system_min_bond_separation),
        system_inter_block_bondsep_(system_inter_block_bondsep),
        system_neighbor_list_(system_neighbor_list),
        block_type_n_atoms_(block_type_n_atoms),
        block_type_atom_types_(block_type_atom_types),
        block_type_n_interblock_bonds_(block_type_n_interblock_bonds),
        block_type_atoms_forming_chemical_bonds_(
            block_type_atoms_forming_chemical_bonds),
        block_type_path_distance_(block_type_path_distance),
        type_params_(type_params),
        global_params_(global_params),
        lj_lk_weights_(lj_lk_weights),
        output_(output) {}

  void calc_energies() override {
    LJRPEDispatch<DeviceDispatch, D, Real, Int>::f(
        context_coords_,
        context_block_type_,
        alternate_coords_,
        alternate_ids_,
        context_system_ids_,
        system_min_bond_separation_,
        system_inter_block_bondsep_,
        system_neighbor_list_,
        block_type_n_atoms_,
        block_type_atom_types_,
        block_type_n_interblock_bonds_,
        block_type_atoms_forming_chemical_bonds_,
        block_type_path_distance_,
        type_params_,
        global_params_,
        lj_lk_weights_,
        output_);
  }

 private:
  TView<Vec<Real, 3>, 3, D> context_coords_;
  TView<Int, 2, D> context_block_type_;
  TView<Vec<Real, 3>, 2, D> alternate_coords_;
  TView<Vec<Int, 3>, 1, D> alternate_ids_;

  TView<Int, 1, D> context_system_ids_;
  TView<Int, 3, D> system_min_bond_separation_;

  TView<Int, 5, D> system_inter_block_bondsep_;

  TView<Int, 3, D> system_neighbor_list_;

  TView<Int, 1, D> block_type_n_atoms_;

  TView<Int, 2, D> block_type_atom_types_;

  TView<Int, 1, D> block_type_n_interblock_bonds_;

  TView<Int, 2, D> block_type_atoms_forming_chemical_bonds_;

  TView<Int, 3, D> block_type_path_distance_;

  // LJ parameters
  TView<LJTypeParams<Real>, 1, D> type_params_;
  TView<LJGlobalParams<Real>, 1, D> global_params_;
  TView<Real, 1, D> lj_lk_weights_;

  TView<Real, 1, D> output_;
};

template <
    template <tmol::Device>
    class DeviceDispatch,
    tmol::Device D,
    typename Real,
    typename Int>
auto LJRPERegistratorDispatch<DeviceDispatch, D, Real, Int>::f(
    TView<Vec<Real, 3>, 3, D> context_coords,
    TView<Int, 2, D> context_block_type,
    TView<Vec<Real, 3>, 2, D> alternate_coords,
    TView<Vec<Int, 3>, 1, D>
        alternate_ids,  // 0 == context id; 1 == block id; 2 == block type

    // which system does a given context belong to
    TView<Int, 1, D> context_system_ids,

    // dims: n-systems x max-n-blocks x max-n-blocks
    // Quick lookup: given the inds of two blocks, ask: what is the minimum
    // number of chemical bonds that separate any pair of atoms in those blocks?
    // If this minimum is greater than the crossover, then no further logic for
    // deciding whether two atoms in those blocks should have their interaction
    // energies calculated: all should. intentionally small to (possibly) fit in
    // constant cache
    TView<Int, 3, D> system_min_bond_separation,

    // dims: n-systems x max-n-blocks x max-n-blocks x
    // max-n-interblock-connections x max-n-interblock-connections
    TView<Int, 5, D> system_inter_block_bondsep,

    // dims n-systems x max-n-blocks x max-n-neighbors
    // -1 as the sentinel
    TView<Int, 3, D> system_neighbor_list,

    //////////////////////
    // Chemical properties
    // how many atoms for a given block
    // Dimsize n_block_types
    TView<Int, 1, D> block_type_n_atoms,

    // what are the atom types for these atoms
    // Dimsize: n_block_types x max_n_atoms
    TView<Int, 2, D> block_type_atom_types,

    // how many inter-block chemical bonds are there
    // Dimsize: n_block_types
    TView<Int, 1, D> block_type_n_interblock_bonds,

    // what atoms form the inter-block chemical bonds
    // Dimsize: n_block_types x max_n_interblock_bonds
    TView<Int, 2, D> block_type_atoms_forming_chemical_bonds,

    // what is the path distance between pairs of atoms in the block
    // Dimsize: n_block_types x max_n_atoms x max_n_atoms
    TView<Int, 3, D> block_type_path_distance,
    //////////////////////

    // LJ parameters
    TView<LJTypeParams<Real>, 1, D> type_params,
    TView<LJGlobalParams<Real>, 1, D> global_params,
    TView<Real, 1, D> lj_lk_weights,
    TView<Real, 1, D> output,
    TView<int64_t, 1, tmol::Device::CPU> annealer) -> void {
  using tmol::pack::sim_anneal::compiled::RPECalc;
  using tmol::pack::sim_anneal::compiled::SimAnnealer;

  int64_t annealer_uint = annealer[0];
  SimAnnealer *sim_annealer = reinterpret_cast<SimAnnealer *>(annealer_uint);
  std::shared_ptr<RPECalc> calc =
      std::make_shared<LJRPECudaCalc<DeviceDispatch, D, Real, Int>>(
          context_coords,
          context_block_type,
          alternate_coords,
          alternate_ids,
          context_system_ids,
          system_min_bond_separation,
          system_inter_block_bondsep,
          system_neighbor_list,
          block_type_n_atoms,
          block_type_atom_types,
          block_type_n_interblock_bonds,
          block_type_atoms_forming_chemical_bonds,
          block_type_path_distance,
          type_params,
          global_params,
          lj_lk_weights,
          output);

  sim_annealer->add_score_component(calc);
}

template struct LJRPEDispatch<ForallDispatch, tmol::Device::CUDA, float, int>;
template struct LJRPEDispatch<ForallDispatch, tmol::Device::CUDA, double, int>;
template struct LJRPERegistratorDispatch<
    ForallDispatch,
    tmol::Device::CUDA,
    float,
    int>;
template struct LJRPERegistratorDispatch<
    ForallDispatch,
    tmol::Device::CUDA,
    double,
    int>;

}  // namespace potentials
}  // namespace ljlk
}  // namespace score
}  // namespace tmol
