#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <Eigen/Geometry>

#include <tmol/utility/tensor/TensorAccessor.h>
#include <tmol/utility/tensor/TensorPack.h>
#include <tmol/utility/tensor/TensorStruct.h>
#include <tmol/utility/tensor/TensorUtil.h>
#include <tmol/utility/nvtx.hh>

#include <tmol/score/common/accumulate.hh>
#include <tmol/score/common/coordinate_load.cuh>
#include <tmol/score/common/count_pair.hh>
#include <tmol/score/common/geom.hh>
#include <tmol/score/common/tuple.hh>

#include <tmol/score/ljlk/potentials/lj.hh>
#include <tmol/score/ljlk/potentials/rotamer_pair_energy_lj.hh>

#include <tmol/pack/sim_anneal/compiled/annealer.hh>

#include <chrono>

#include <tmol/score/common/forall_dispatch.cuda.impl.cuh>

#include <moderngpu/cta_load_balance.hxx>
#include <moderngpu/cta_reduce.hxx>
#include <moderngpu/cta_scan.hxx>
#include <moderngpu/cta_segreduce.hxx>
#include <moderngpu/cta_segscan.hxx>
#include <moderngpu/memory.hxx>
#include <moderngpu/search.hxx>
#include <moderngpu/transform.hxx>

// This file moves in more recent versions of Torch
#include <ATen/cuda/CUDAStream.h>

// #include <tmol/score/ljlk/potentials/rotamer_pair_energy_lj.impl.hh>

// The maximum number of inter-residue chemical bonds
#define MAX_N_CONN 4
#define TILE_SIZE 32

namespace tmol {
namespace score {
namespace ljlk {
namespace potentials {

static int already_printed = 0;

template <typename Real, int N>
using Vec = Eigen::Matrix<Real, N, 1>;

template <
    template <tmol::Device>
    class DeviceDispatch,
    tmol::Device D,
    typename Real,
    typename Int>
auto LJRPEDispatch<DeviceDispatch, D, Real, Int>::f(
    TView<Vec<Real, 3>, 3, D> context_coords,
    TView<Int, 2, D> context_block_type,
    TView<Vec<Real, 3>, 2, D> alternate_coords,
    TView<Vec<Int, 3>, 1, D>
        alternate_ids,  // 0 == context id; 1 == block id; 2 == block type

    // which system does a given context belong to
    TView<Int, 1, D> context_system_ids,

    // dims: n-systems x max-n-blocks x max-n-blocks
    // Quick lookup: given the inds of two blocks, ask: what is the minimum
    // number of chemical bonds that separate any pair of atoms in those blocks?
    // If this minimum is greater than the crossover, then no further logic for
    // deciding whether two atoms in those blocks should have their interaction
    // energies calculated: all should. intentionally small to (possibly) fit in
    // constant cache
    TView<Int, 3, D> system_min_bond_separation,

    // dims: n-systems x max-n-blocks x max-n-blocks x
    // max-n-interblock-connections x max-n-interblock-connections
    TView<Int, 5, D> system_inter_block_bondsep,

    // dims n-systems x max-n-blocks x max-n-neighbors
    // -1 as the sentinel
    TView<Int, 3, D> system_neighbor_list,

    //////////////////////
    // Chemical properties
    // how many atoms for a given block
    // Dimsize n_block_types
    TView<Int, 1, D> block_type_n_atoms,

    // what are the atom types for these atoms
    // Dimsize: n_block_types x max_n_atoms
    TView<Int, 2, D> block_type_atom_types,

    // how many inter-block chemical bonds are there
    // Dimsize: n_block_types
    TView<Int, 1, D> block_type_n_interblock_bonds,

    // what atoms form the inter-block chemical bonds
    // Dimsize: n_block_types x max_n_interblock_bonds
    TView<Int, 2, D> block_type_atoms_forming_chemical_bonds,

    // what is the path distance between pairs of atoms in the block
    // Dimsize: n_block_types x max_n_atoms x max_n_atoms
    TView<Int, 3, D> block_type_path_distance,
    //////////////////////

    // LJ parameters
    TView<LJTypeParams<Real>, 1, D> type_params,
    TView<LJGlobalParams<Real>, 1, D> global_params,
    TView<Real, 1, D> lj_lk_weights,
    TView<Real, 1, D> output) -> void {
  int const n_systems = system_min_bond_separation.size(0);
  int const n_contexts = context_coords.size(0);
  int64_t const n_alternate_blocks = alternate_coords.size(0);
  int const max_n_blocks = context_coords.size(1);
  int64_t const max_n_atoms = context_coords.size(2);
  int const n_block_types = block_type_n_atoms.size(0);
  int const max_n_interblock_bonds =
      block_type_atoms_forming_chemical_bonds.size(1);
  int64_t const max_n_neighbors = system_neighbor_list.size(2);

  assert(alternate_coords.size(1) == max_n_atoms);
  assert(alternate_ids.size(0) == n_alternate_blocks);
  assert(context_coords.size(0) == context_block_type.size(0));
  assert(context_system_ids.size(0) == n_contexts);

  assert(system_min_bond_separation.size(1) == max_n_blocks);
  assert(system_min_bond_separation.size(2) == max_n_blocks);

  assert(system_inter_block_bondsep.size(0) == n_systems);
  assert(system_inter_block_bondsep.size(1) == max_n_blocks);
  assert(system_inter_block_bondsep.size(2) == max_n_blocks);
  assert(system_inter_block_bondsep.size(3) == max_n_interblock_bonds);
  assert(system_inter_block_bondsep.size(4) == max_n_interblock_bonds);
  assert(system_neighbor_list.size(0) == n_systems);
  assert(system_neighbor_list.size(1) == max_n_blocks);

  assert(block_type_atom_types.size(0) == n_block_types);
  assert(block_type_atom_types.size(1) == max_n_atoms);
  assert(block_type_n_interblock_bonds.size(0) == n_block_types);
  assert(block_type_atoms_forming_chemical_bonds.size(0) == n_block_types);
  assert(block_type_path_distance.size(0) == n_block_types);
  assert(block_type_path_distance.size(1) == max_n_atoms);
  assert(block_type_path_distance.size(2) == max_n_atoms);

  assert(lj_lk_weights.size(0) == 2);

  // auto wcts = std::chrono::system_clock::now();
  // clock_t start_time = clock();

  // Allocate and zero the output tensors in a separate stream

  // auto output_t = TPack<Real, 1, D>::zeros({n_alternate_blocks});
  // auto output = output_t.view;
  // auto count_t = TPack<int, 1, D>::zeros({1});
  // auto count = count_t.view;
  //
  // // I'm not sure I want/need events for synchronization
  // auto event_t = TPack<int64_t, 1, D>::zeros({2});

  // return {output_t, event_t};

  using namespace mgpu;
  typedef launch_box_t<
      arch_20_cta<64, 5>,
      arch_35_cta<64, 5>,
      arch_52_cta<64, 5>>
      launch_t;

  // between one alternate rotamer and its neighbors in the surrounding context
  auto score_inter_pairs = ([=] MGPU_DEVICE(
                                int tid,
                                int alt_start_atom,
                                int neighb_start_atom,
                                Real *alt_coords,
                                Real *neighb_coords,
                                LJTypeParams<Real> *alt_params,
                                LJTypeParams<Real> *neighb_params,
                                int const max_important_bond_separation,
                                int const alt_block_ind,
                                int const neighb_block_ind,
                                int const alt_block_type,
                                int const neighb_block_type,

                                int min_separation,
                                TensorAccessor<Int, 4, D> inter_block_bondsep,

                                int const alt_n_atoms,
                                int const neighb_n_atoms,
                                int const n_conn1,
                                int const n_conn2,
                                int const *path_dist1,
                                int const *path_dist2,
                                int const *conn_seps) {
    Real score_total = 0;
    Real coord1[3];
    Real coord2[3];

    int const alt_remain = min(TILE_SIZE, alt_n_atoms - alt_start_atom);
    int const neighb_remain =
        min(TILE_SIZE, neighb_n_atoms - neighb_start_atom);

    int const n_pairs = alt_remain * neighb_remain;

    LJGlobalParams<Real> global_params_local = global_params[0];
    Real lj_weight = lj_lk_weights[0];

    for (int i = tid; i < n_pairs; i += blockDim.x) {
      int const alt_atom_tile_ind = i / neighb_remain;
      int const neighb_atom_tile_ind = i % neighb_remain;
      int const alt_atom_ind = alt_atom_tile_ind + alt_start_atom;
      int const neighb_atom_ind = neighb_atom_tile_ind + neighb_start_atom;
      for (int j = 0; j < 3; ++j) {
        coord1[j] = alt_coords[3 * alt_atom_tile_ind + j];
        coord2[j] = neighb_coords[3 * neighb_atom_tile_ind + j];
      }

      // int const separation = 5;
      Real dist2 =
          ((coord1[0] - coord2[0]) * (coord1[0] - coord2[0])
           + (coord1[1] - coord2[1]) * (coord1[1] - coord2[1])
           + (coord1[2] - coord2[2]) * (coord1[2] - coord2[2]));
      if (dist2 > 36.0) {
        // DANGER -- maximum reach of LJ potential hard coded here in a second
        // place out of range!
        continue;
      }
      Real dist = std::sqrt(dist2);

      int separation = min_separation;
      if (separation <= max_important_bond_separation) {
        separation =
            common::count_pair::CountPair<D, Int>::inter_block_separation<
                TILE_SIZE>(
                max_important_bond_separation,
                alt_atom_tile_ind,
                neighb_atom_tile_ind,
                n_conn1,
                n_conn2,
                path_dist1,
                path_dist2,
                conn_seps);
      }
      // if (separation != separation2){
      // 	printf("separation mismatch! %d %d %d %d %d\n", alt_atom_ind,
      // neighb_atom_ind, min_separation, separation, separation2);
      // }

      // TEMP short circuit the lennard-jones evaluation
      // Real lj = separation > 5 ? dist : 0;

      Real lj = lj_score<Real>::V(
          dist,
          separation,
          alt_params[alt_atom_tile_ind],
          neighb_params[neighb_atom_tile_ind],
          global_params_local);
      lj *= lj_weight;

      // if ( lj != 0 ) {
      //   printf("cuda %d %d %6.3f %6.3f %6.3f vs %6.3f %6.3f %6.3f e=
      //   %8.4f\n",
      //     alt_atom_ind, neighb_atom_ind,
      //     coord1[0], coord1[1], coord1[2],
      //     coord2[0], coord2[1], coord2[2],
      //     lj
      //   );
      // }

      score_total += lj;
    }
    return score_total;
  });

  // between one atoms within an alternate rotamer
  auto score_intra_pairs = ([=] MGPU_DEVICE(
                                int tid,
                                int start_atom1,
                                int start_atom2,
                                Real *coords1,
                                Real *coords2,
                                LJTypeParams<Real> *params1,
                                LJTypeParams<Real> *params2,
                                int const max_important_bond_separation,
                                int const block_type,
                                int const n_atoms) {
    Real score_total = 0;
    Real coord1[3];
    Real coord2[3];

    int const remain1 = min(TILE_SIZE, n_atoms - start_atom1);
    int const remain2 = min(TILE_SIZE, n_atoms - start_atom2);

    int const n_pairs = remain1 * remain2;

    LJGlobalParams<Real> global_params_local = global_params[0];
    Real lj_weight = lj_lk_weights[0];

    for (int i = tid; i < n_pairs; i += blockDim.x) {
      int const atom_ind_1_local = i / remain2;
      int const atom_ind_2_local = i % remain2;
      int const atom_ind_1 = atom_ind_1_local + start_atom1;
      int const atom_ind_2 = atom_ind_2_local + start_atom2;
      if (atom_ind_1 >= atom_ind_2) {
        continue;
      }

      for (int j = 0; j < 3; ++j) {
        coord1[j] = coords1[3 * atom_ind_1_local + j];
        coord2[j] = coords2[3 * atom_ind_2_local + j];
      }
      // int const atom_1_type = atom_type1[atom_ind_1_local];
      // int const atom_2_type = atom_type2[atom_ind_2_local];

      int const separation =
          block_type_path_distance[block_type][atom_ind_1][atom_ind_2];

      Real const dist = sqrt(
          (coord1[0] - coord2[0]) * (coord1[0] - coord2[0])
          + (coord1[1] - coord2[1]) * (coord1[1] - coord2[1])
          + (coord1[2] - coord2[2]) * (coord1[2] - coord2[2]));

      Real lj = lj_score<Real>::V(
          dist,
          separation,
          params1[atom_ind_1_local],
          params2[atom_ind_2_local],
          global_params_local);
      lj *= lj_lk_weights[0];
      score_total += lj;
    }
    return score_total;
  });

  auto eval_energies = ([=] MGPU_DEVICE(int tid, int cta) {
    typedef typename launch_t::sm_ptx params_t;
    enum {
      nt = params_t::nt,
      vt = params_t::vt,
      vt0 = params_t::vt0,
      nv = nt * vt
    };
    typedef mgpu::cta_reduce_t<nt, Real> reduce_t;

    struct struct_part1 {
      Real coords_alt1[TILE_SIZE * 3];  // 786 bytes for coords
      Real coords_alt2[TILE_SIZE * 3];
      LJTypeParams<Real> params_alt1[TILE_SIZE];  // 1536 bytes for params
      LJTypeParams<Real> params_alt2[TILE_SIZE];
      Int min_separation;  // 8 bytes for two integers
      Int n_conn_alt;
      Int conn_ats_alt1[MAX_N_CONN];  // 32 bytes for conn ats
      Int conn_ats_alt2[MAX_N_CONN];
      Int path_dist_alt1[MAX_N_CONN * TILE_SIZE];  // 1024 for path dists
      Int path_dist_alt2[MAX_N_CONN * TILE_SIZE];
    };

    __shared__ struct shared_mem_struct {
      Real coords_alt1[TILE_SIZE * 3];  // 786 bytes for coords
      Real coords_alt2[TILE_SIZE * 3];
      LJTypeParams<Real> params_alt1[TILE_SIZE];  // 1536 bytes for params
      LJTypeParams<Real> params_alt2[TILE_SIZE];
      Int min_separation;  // 8 bytes for two integers
      Int n_conn_alt;
      Int conn_ats_alt1[MAX_N_CONN];  // TILE_SIZE bytes for conn ats
      Int conn_ats_alt2[MAX_N_CONN];
      Int path_dist_alt1[MAX_N_CONN * TILE_SIZE];  // 1024 for path dists
      Int path_dist_alt2[MAX_N_CONN * TILE_SIZE];

      union union_pt2_red {
        struct struct_part2 {
          Real coords_other[TILE_SIZE * 3];             // 384 bytes for coords
          Int n_conn_other;                             // 4 bytes for an int
          LJTypeParams<Real> params_other[TILE_SIZE];   // 768 bytes for params
          Int conn_ats_other[MAX_N_CONN];               // 16 bytes
          Int path_dist_other[MAX_N_CONN * TILE_SIZE];  // 512 bypes
          Int conn_seps[MAX_N_CONN * MAX_N_CONN];  // 64 bytes for conn/conn
        } vals;

      } union_vals;
      typename reduce_t::storage_t reduce;
    } shared;

    if (false) {
      // if (cta == 0 and tid == 0) {
      printf(
          "sizeof shared_mem_struct %lu, reduce size %lu, pt1 %lu, pt2 %lu, "
          "union %lu\n",
          sizeof(shared_mem_struct),
          sizeof(reduce_t::storage_t),
          sizeof(struct_part1),
          sizeof(shared_mem_struct::union_pt2_red::struct_part2),
          sizeof(shared_mem_struct::union_pt2_red));
    }

    Real *coords_alt1 = shared.coords_alt1;
    Real *coords_alt2 = shared.coords_alt2;
    Real *coords_other = shared.union_vals.vals.coords_other;
    LJTypeParams<Real> *params_alt1 = shared.params_alt1;
    LJTypeParams<Real> *params_alt2 = shared.params_alt2;
    LJTypeParams<Real> *params_other = shared.union_vals.vals.params_other;

    Int last_alt_ind = -1;
    bool count_pair_data_loaded = false;

    for (int iteration = 0; iteration < vt; ++iteration) {
      Real totalE1 = 0;
      Real totalE2 = 0;

      int alt_ind = (vt * cta + iteration) / max_n_neighbors;

      if (alt_ind >= n_alternate_blocks / 2) {
        break;
      }
      bool const new_alt = alt_ind != last_alt_ind;
      // last_alt_ind = alt_ind;
      if (new_alt) {
        count_pair_data_loaded = false;
      }

      int neighb_ind = (vt * cta + iteration) % max_n_neighbors;

      int const max_important_bond_separation = 4;
      int const alt_context = alternate_ids[2 * alt_ind][0];
      if (alt_context == -1) {
        continue;
      }

      int const alt_block_ind = alternate_ids[2 * alt_ind][1];
      int const alt_block_type1 = alternate_ids[2 * alt_ind][2];
      int const alt_block_type2 = alternate_ids[2 * alt_ind + 1][2];
      // if (tid == 0) {
      // 	printf("alt block type: %d ind, %d type1, %d ind type2\n",
      // alt_ind, alt_block_type1, alt_block_type2);
      // }
      int const system = context_system_ids[alt_context];
      int const alt_n_atoms1 = block_type_n_atoms[alt_block_type1];
      int const alt_n_atoms2 = block_type_n_atoms[alt_block_type2];

      int const neighb_block_ind =
          system_neighbor_list[system][alt_block_ind][neighb_ind];
      if (neighb_block_ind == -1) {
        continue;
      }

      if (alt_block_ind != neighb_block_ind) {
        int const neighb_block_type =
            context_block_type[alt_context][neighb_block_ind];
        int const neighb_n_atoms = block_type_n_atoms[neighb_block_type];

        if (tid == 0) {
          int const min_sep = system_min_bond_separation[system][alt_block_ind]
                                                        [neighb_block_ind];
          // printf("min_sep %2d\n", min_sep);
          shared.min_separation = min_sep;
          int const n_conn_alt = block_type_n_interblock_bonds[alt_block_type1];
          int const n_conn_other =
              block_type_n_interblock_bonds[neighb_block_type];
          shared.n_conn_alt = n_conn_alt;
          shared.union_vals.vals.n_conn_other = n_conn_other;
        }
        __syncthreads();

        int const min_sep = shared.min_separation;

        bool const count_pair_striking_dist =
            min_sep <= max_important_bond_separation;

        int const n_conn_alt = shared.n_conn_alt;
        int const n_conn_other = shared.union_vals.vals.n_conn_other;
        if (count_pair_striking_dist && tid < n_conn_alt) {
          shared.conn_ats_alt1[tid] =
              block_type_atoms_forming_chemical_bonds[alt_block_type1][tid];
          shared.conn_ats_alt2[tid] =
              block_type_atoms_forming_chemical_bonds[alt_block_type2][tid];
        }
        if (count_pair_striking_dist && tid < n_conn_other) {
          shared.union_vals.vals.conn_ats_other[tid] =
              block_type_atoms_forming_chemical_bonds[neighb_block_type][tid];
        }
        if (count_pair_striking_dist && tid < n_conn_alt * n_conn_other) {
          int conn1 = tid / n_conn_other;
          int conn2 = tid % n_conn_other;
          shared.union_vals.vals.conn_seps[tid] =
              system_inter_block_bondsep[system][alt_block_ind]
                                        [neighb_block_ind][conn1][conn2];
        }
        __syncthreads();

        // Tile the sets of TILE_SIZE atoms
        int const alt_n_iterations =
            (max(alt_n_atoms1, alt_n_atoms2) - 4 - 1) / TILE_SIZE + 1;
        int const neighb_n_iterations =
            (neighb_n_atoms - 4 - 1) / TILE_SIZE + 1;

        for (int i = 0; i < alt_n_iterations; ++i) {
          if (i != 0) {
            // make sure all threads have completed their work
            // from the previous iteration before we overwrite
            // the contents of shared memory
            __syncthreads();
          }

          // Let's load coordinates and Lennard-Jones parameters for
          // TILE_SIZE atoms into shared memory
          int const i_n_atoms_to_load1 =
              min(Int(TILE_SIZE), Int((alt_n_atoms1 - TILE_SIZE * i - 4)));

          int const i_n_atoms_to_load2 =
              min(Int(TILE_SIZE), Int((alt_n_atoms2 - TILE_SIZE * i - 4)));

          if (new_alt || alt_n_atoms1 > TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind][4]),
                tid,
                i_n_atoms_to_load1 * 3,
                coords_alt1,
                false);
          }

          if (new_alt || alt_n_atoms2 > TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind + 1][4]),
                tid,
                i_n_atoms_to_load2 * 3,
                coords_alt2,
                false);
          }

          // continue; //  GOOD!

          if ((new_alt || alt_n_atoms1 > TILE_SIZE) && tid < TILE_SIZE) {
            // coalesced read of atom coordinate data
            // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
            //     alternate_coords[2 * alt_ind], i * TILE_SIZE + 4,
            //     coords_alt1, tid);

            // load the Lennard-Jones parameters for these TILE_SIZE atoms
            if (tid < i_n_atoms_to_load1) {
              int const atid = TILE_SIZE * i + tid + 4;
              int const attype = block_type_atom_types[alt_block_type1][atid];
              if (attype >= 0) {
                params_alt1[tid] = type_params[attype];
              }
            }
          }

          if ((new_alt || alt_n_atoms1 > TILE_SIZE || !count_pair_data_loaded)
              && tid < i_n_atoms_to_load1) {
            int const atid = TILE_SIZE * i + tid + 4;
            if (count_pair_striking_dist && !count_pair_data_loaded) {
              for (int j = 0; j < n_conn_alt; ++j) {
                int ij_path_dist =
                    block_type_path_distance[alt_block_type1]
                                            [shared.conn_ats_alt1[j]][atid];
                shared.path_dist_alt1[j * TILE_SIZE + tid] = ij_path_dist;
              }
            }
          }

          // continue; // GOOD

          if ((new_alt || alt_n_atoms2 > TILE_SIZE)
              && tid < i_n_atoms_to_load2) {
            // load the Lennard-Jones parameters for these TILE_SIZE atoms
            int const atid = TILE_SIZE * i + tid + 4;
            int const attype = block_type_atom_types[alt_block_type2][atid];
            if (attype >= 0) {
              params_alt2[tid] = type_params[attype];
            }
          }

          // continue; // BAD??!!

          if ((new_alt || alt_n_atoms2 > TILE_SIZE || !count_pair_data_loaded)
              && tid < i_n_atoms_to_load2) {
            int const atid = TILE_SIZE * i + tid + 4;
            if (count_pair_striking_dist && !count_pair_data_loaded) {
              for (int j = 0; j < n_conn_alt; ++j) {
                int ij_path_dist =
                    block_type_path_distance[alt_block_type2]
                                            [shared.conn_ats_alt2[j]][atid];
                shared.path_dist_alt2[j * TILE_SIZE + tid] = ij_path_dist;
              }
            }
          }
          if (count_pair_striking_dist) {
            count_pair_data_loaded = true;
          }
          // continue; // BAD

          for (int j = 0; j < neighb_n_iterations; ++j) {
            if (j != 0) {
              // make sure that all threads have finished energy
              // calculations from the previous iteration
              __syncthreads();
            }
            int j_n_atoms_to_load =
                min(Int(TILE_SIZE), Int((neighb_n_atoms - TILE_SIZE * j - 4)));
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(
                    &context_coords[alt_context][neighb_block_ind][4]),
                tid,
                j_n_atoms_to_load * 3,
                coords_other,
                false);

            if (tid < TILE_SIZE) {
              // Coalesced read of atom coordinate data
              // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
              //     context_coords[alt_context][neighb_block_ind],
              //     j * TILE_SIZE + 4,
              //     coords_other,
              //     tid);

              // load the Lennard-Jones parameters for these TILE_SIZE atoms
              if (tid < j_n_atoms_to_load) {
                int const atid = TILE_SIZE * j + 4 + tid;
                int const attype =
                    block_type_atom_types[neighb_block_type][atid];
                if (attype >= 0) {
                  params_other[tid] = type_params[attype];
                }
                if (count_pair_striking_dist) {
                  for (int k = 0; k < n_conn_other; ++k) {
                    int jk_path_dist =
                        block_type_path_distance[neighb_block_type]
                                                [shared.union_vals.vals
                                                     .conn_ats_other[k]][atid];
                    shared.union_vals.vals
                        .path_dist_other[k * TILE_SIZE + tid] = jk_path_dist;
                  }
                }
              }
            }

            // make sure shared-memory loading has completed before we proceed
            // into energy calculations
            __syncthreads();

            // Now we will calculate the TILE_SIZExTILE_SIZE atom pair energies
            totalE1 = score_inter_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt1,
                coords_other,
                params_alt1,
                params_other,
                max_important_bond_separation,
                alt_block_ind,
                neighb_block_ind,
                alt_block_type1,
                neighb_block_type,
                min_sep,
                system_inter_block_bondsep[system],
                alt_n_atoms1,
                neighb_n_atoms,
                n_conn_alt,
                n_conn_other,
                shared.path_dist_alt1,
                shared.union_vals.vals.path_dist_other,
                shared.union_vals.vals.conn_seps);

            totalE2 = score_inter_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt2,
                coords_other,
                params_alt2,
                params_other,
                max_important_bond_separation,
                alt_block_ind,
                neighb_block_ind,
                alt_block_type2,
                neighb_block_type,
                min_sep,
                system_inter_block_bondsep[system],
                alt_n_atoms2,
                neighb_n_atoms,
                n_conn_alt,
                n_conn_other,
                shared.path_dist_alt2,
                shared.union_vals.vals.path_dist_other,
                shared.union_vals.vals.conn_seps);
          }  // for j
        }    // for i
      } else {
        // int const alt_n_atoms = block_type_n_atoms[alt_block_type];

        int const n_iterations =
            (max(alt_n_atoms1, alt_n_atoms2) - 4 - 1) / TILE_SIZE + 1;

        for (int i = 0; i < n_iterations; ++i) {
          if (i != 0) {
            // make sure the calculations for the previous iteration
            // have completed before we overwrite the contents of
            // shared memory
            __syncthreads();
          }
          int const i_n_atoms_to_load1 =
              min(Int(TILE_SIZE), Int((alt_n_atoms1 - TILE_SIZE * i - 4)));

          int const i_n_atoms_to_load2 =
              min(Int(TILE_SIZE), Int((alt_n_atoms2 - TILE_SIZE * i - 4)));

          if ((new_alt || alt_n_atoms1 > TILE_SIZE) && tid < TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind][4]),
                tid,
                i_n_atoms_to_load1 * 3,
                coords_alt1,
                false);

            // load Lennard-Jones parameters for the TILE_SIZE atoms into shared
            // memory
            if (i * TILE_SIZE + 4 + tid < max_n_atoms) {
              int const atind = i * TILE_SIZE + tid + 4;
              int const attype = block_type_atom_types[alt_block_type1][atind];
              if (attype >= 0) {
                params_alt1[tid] = type_params[attype];
              }
            }
          }
          if ((new_alt || alt_n_atoms2 > TILE_SIZE) && tid < TILE_SIZE) {
            mgpu::mem_to_shared<TILE_SIZE, 3>(
                reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind + 1][4]),
                tid,
                i_n_atoms_to_load2 * 3,
                coords_alt2,
                false);
            // coalesced reads of coordinate data
            // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
            //     alternate_coords[2 * alt_ind + 1],
            //     i * TILE_SIZE + 4,
            //     coords_alt2,
            //     tid);

            // load Lennard-Jones parameters for the TILE_SIZE atoms into shared
            // memory
            if (i * TILE_SIZE + 4 + tid < max_n_atoms) {
              int const atind = i * TILE_SIZE + tid + 4;
              int const attype = block_type_atom_types[alt_block_type2][atind];
              if (attype >= 0) {
                params_alt2[tid] = type_params[attype];
              }
            }
          }

          // process residue 1
          for (int j = i; j < n_iterations; ++j) {
            if (j != i) {
              // make sure calculations from the previous iteration have
              // completed before we overwrite the contents of shared
              // memory
              __syncthreads();
            }

            if (j != i && tid < TILE_SIZE) {
              mgpu::mem_to_shared<TILE_SIZE, 3>(
                  reinterpret_cast<Real *>(&alternate_coords[2 * alt_ind][4]),
                  tid,
                  i_n_atoms_to_load1 * 3,
                  coords_other,
                  false);
              // coalesced read of coordinate data
              // common::coalesced_read_of_TILE_SIZE_coords_into_shared(
              //     alternate_coords[2 * alt_ind], j * TILE_SIZE + 4,
              //     coords_other, tid);
              if (j * TILE_SIZE + tid < max_n_atoms) {
                int const atind = j * TILE_SIZE + 4 + tid;
                int const attype =
                    block_type_atom_types[alt_block_type1][atind];
                if (attype >= 0) {
                  params_other[tid] = type_params[attype];
                }
              }
            }
            __syncthreads();
            totalE1 = score_intra_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt1,
                (i == j ? coords_alt1 : coords_other),
                params_alt1,
                (i == j ? params_alt1 : params_other),
                max_important_bond_separation,
                alt_block_type1,
                alt_n_atoms1);
          }  // for j

          // Process residue 2
          for (int j = i; j < n_iterations; ++j) {
            if (j != i) {
              // make sure calculations from the previous iteration have
              // completed before we overwrite the contents of shared
              // memory
              __syncthreads();
            }

            if (j != i && tid < TILE_SIZE) {
              mgpu::mem_to_shared<TILE_SIZE, 3>(
                  reinterpret_cast<Real *>(
                      &alternate_coords[2 * alt_ind + 1][4]),
                  tid,
                  i_n_atoms_to_load2 * 3,
                  coords_other,
                  false);
              if (j * TILE_SIZE + tid < max_n_atoms) {
                int const atind = j * TILE_SIZE + 4 + tid;
                int const attype =
                    block_type_atom_types[alt_block_type2][atind];
                if (attype >= 0) {
                  params_other[tid] = type_params[attype];
                }
              }
            }
            __syncthreads();
            totalE2 = score_intra_pairs(
                tid,
                i * TILE_SIZE + 4,
                j * TILE_SIZE + 4,
                coords_alt2,
                (i == j ? coords_alt2 : coords_other),
                params_alt2,
                (i == j ? params_alt2 : params_other),
                max_important_bond_separation,
                alt_block_type2,
                alt_n_atoms2);
          }  // for j
        }    // for i
      }      // else

      __syncthreads();

      Real const cta_totalE1 = reduce_t().reduce(
          tid, totalE1, shared.reduce, nt, mgpu::plus_t<Real>());

      Real const cta_totalE2 = reduce_t().reduce(
          tid, totalE2, shared.reduce, nt, mgpu::plus_t<Real>());

      if (tid == 0) {
        // printf("%d %d %f; %d %d %f\n", 2 * alt_ind, neighb_ind, cta_totalE1,
        // 2 * alt_ind + 1, neighb_ind, cta_totalE2);
        atomicAdd(&output[2 * alt_ind], cta_totalE1);
        atomicAdd(&output[2 * alt_ind + 1], cta_totalE2);
      }
    }
  });

  at::cuda::CUDAStream wrapped_stream = at::cuda::getStreamFromPool();
  setCurrentCUDAStream(wrapped_stream);
  mgpu::standard_context_t context(wrapped_stream.stream());

  // mgpu::standard_context_t context;

  int const n_ctas =
      (n_alternate_blocks * max_n_neighbors / 2 - 1) / launch_t::sm_ptx::vt + 1;
  if (already_printed == 0) {
    std::cout << "n_ctas: " << n_ctas << " n_alternate_blocks "
              << n_alternate_blocks << " max_n_neighbors " << max_n_neighbors
              << std::endl;
    already_printed = 1;
  }
  mgpu::cta_launch<launch_t>(eval_energies, n_ctas, context);

  at::cuda::setCurrentCUDAStream(at::cuda::getDefaultCUDAStream());

#ifdef __HIPCC__
  // float first;
  // hipMemcpy(&first, &output[0], sizeof(float), hipMemcpyDeviceToHost);
  //
  // clock_t stop_time = clock();
  // std::chrono::duration<double> wctduration =
  // (std::chrono::system_clock::now() - wcts);
  //
  // std::cout << n_systems << " " << n_contexts << " " <<n_alternate_blocks <<
  // " "; std::cout << n_alternate_blocks * max_n_neighbors * max_n_atoms *
  // max_n_atoms << " "; std::cout << "runtime? " << ((double)stop_time -
  // start_time) / CLOCKS_PER_SEC
  //           << " wall time: " << wctduration.count() << " " << first
  //           << std::endl;
#endif
  // return {output_t, event_t};
}

template <
    template <tmol::Device>
    class DeviceDispatch,
    tmol::Device D,
    typename Real,
    typename Int>
class LJRPECudaCalc : public pack::sim_anneal::compiled::RPECalc {
 public:
  LJRPECudaCalc(
      TView<Vec<Real, 3>, 3, D> context_coords,
      TView<Int, 2, D> context_block_type,
      TView<Vec<Real, 3>, 2, D> alternate_coords,
      TView<Vec<Int, 3>, 1, D>
          alternate_ids,  // 0 == context id; 1 == block id; 2 == block type

      // which system does a given context belong to
      TView<Int, 1, D> context_system_ids,

      // dims: n-systems x max-n-blocks x max-n-blocks
      // Quick lookup: given the inds of two blocks, ask: what is the minimum
      // number of chemical bonds that separate any pair of atoms in those
      // blocks? If this minimum is greater than the crossover, then no further
      // logic for deciding whether two atoms in those blocks should have their
      // interaction energies calculated: all should. intentionally small to
      // (possibly) fit in constant cache
      TView<Int, 3, D> system_min_bond_separation,

      // dims: n-systems x max-n-blocks x max-n-blocks x
      // max-n-interblock-connections x max-n-interblock-connections
      TView<Int, 5, D> system_inter_block_bondsep,

      // dims n-systems x max-n-blocks x max-n-neighbors
      // -1 as the sentinel
      TView<Int, 3, D> system_neighbor_list,

      //////////////////////
      // Chemical properties
      // how many atoms for a given block
      // Dimsize n_block_types
      TView<Int, 1, D> block_type_n_atoms,

      // what are the atom types for these atoms
      // Dimsize: n_block_types x max_n_atoms
      TView<Int, 2, D> block_type_atom_types,

      // how many inter-block chemical bonds are there
      // Dimsize: n_block_types
      TView<Int, 1, D> block_type_n_interblock_bonds,

      // what atoms form the inter-block chemical bonds
      // Dimsize: n_block_types x max_n_interblock_bonds
      TView<Int, 2, D> block_type_atoms_forming_chemical_bonds,

      // what is the path distance between pairs of atoms in the block
      // Dimsize: n_block_types x max_n_atoms x max_n_atoms
      TView<Int, 3, D> block_type_path_distance,
      //////////////////////

      // LJ parameters
      TView<LJTypeParams<Real>, 1, D> type_params,
      TView<LJGlobalParams<Real>, 1, D> global_params,
      TView<Real, 1, D> lj_lk_weights,
      TView<Real, 1, D> output)
      : context_coords_(context_coords),
        context_block_type_(context_block_type),
        alternate_coords_(alternate_coords),
        alternate_ids_(alternate_ids),
        context_system_ids_(context_system_ids),
        system_min_bond_separation_(system_min_bond_separation),
        system_inter_block_bondsep_(system_inter_block_bondsep),
        system_neighbor_list_(system_neighbor_list),
        block_type_n_atoms_(block_type_n_atoms),
        block_type_atom_types_(block_type_atom_types),
        block_type_n_interblock_bonds_(block_type_n_interblock_bonds),
        block_type_atoms_forming_chemical_bonds_(
            block_type_atoms_forming_chemical_bonds),
        block_type_path_distance_(block_type_path_distance),
        type_params_(type_params),
        global_params_(global_params),
        lj_lk_weights_(lj_lk_weights),
        output_(output) {}

  void calc_energies() override {
    LJRPEDispatch<DeviceDispatch, D, Real, Int>::f(
        context_coords_,
        context_block_type_,
        alternate_coords_,
        alternate_ids_,
        context_system_ids_,
        system_min_bond_separation_,
        system_inter_block_bondsep_,
        system_neighbor_list_,
        block_type_n_atoms_,
        block_type_atom_types_,
        block_type_n_interblock_bonds_,
        block_type_atoms_forming_chemical_bonds_,
        block_type_path_distance_,
        type_params_,
        global_params_,
        lj_lk_weights_,
        output_);
  }

 private:
  TView<Vec<Real, 3>, 3, D> context_coords_;
  TView<Int, 2, D> context_block_type_;
  TView<Vec<Real, 3>, 2, D> alternate_coords_;
  TView<Vec<Int, 3>, 1, D> alternate_ids_;

  TView<Int, 1, D> context_system_ids_;
  TView<Int, 3, D> system_min_bond_separation_;

  TView<Int, 5, D> system_inter_block_bondsep_;

  TView<Int, 3, D> system_neighbor_list_;

  TView<Int, 1, D> block_type_n_atoms_;

  TView<Int, 2, D> block_type_atom_types_;

  TView<Int, 1, D> block_type_n_interblock_bonds_;

  TView<Int, 2, D> block_type_atoms_forming_chemical_bonds_;

  TView<Int, 3, D> block_type_path_distance_;

  // LJ parameters
  TView<LJTypeParams<Real>, 1, D> type_params_;
  TView<LJGlobalParams<Real>, 1, D> global_params_;
  TView<Real, 1, D> lj_lk_weights_;

  TView<Real, 1, D> output_;
};

template <
    template <tmol::Device>
    class DeviceDispatch,
    tmol::Device D,
    typename Real,
    typename Int>
auto LJRPERegistratorDispatch<DeviceDispatch, D, Real, Int>::f(
    TView<Vec<Real, 3>, 3, D> context_coords,
    TView<Int, 2, D> context_block_type,
    TView<Vec<Real, 3>, 2, D> alternate_coords,
    TView<Vec<Int, 3>, 1, D>
        alternate_ids,  // 0 == context id; 1 == block id; 2 == block type

    // which system does a given context belong to
    TView<Int, 1, D> context_system_ids,

    // dims: n-systems x max-n-blocks x max-n-blocks
    // Quick lookup: given the inds of two blocks, ask: what is the minimum
    // number of chemical bonds that separate any pair of atoms in those blocks?
    // If this minimum is greater than the crossover, then no further logic for
    // deciding whether two atoms in those blocks should have their interaction
    // energies calculated: all should. intentionally small to (possibly) fit in
    // constant cache
    TView<Int, 3, D> system_min_bond_separation,

    // dims: n-systems x max-n-blocks x max-n-blocks x
    // max-n-interblock-connections x max-n-interblock-connections
    TView<Int, 5, D> system_inter_block_bondsep,

    // dims n-systems x max-n-blocks x max-n-neighbors
    // -1 as the sentinel
    TView<Int, 3, D> system_neighbor_list,

    //////////////////////
    // Chemical properties
    // how many atoms for a given block
    // Dimsize n_block_types
    TView<Int, 1, D> block_type_n_atoms,

    // what are the atom types for these atoms
    // Dimsize: n_block_types x max_n_atoms
    TView<Int, 2, D> block_type_atom_types,

    // how many inter-block chemical bonds are there
    // Dimsize: n_block_types
    TView<Int, 1, D> block_type_n_interblock_bonds,

    // what atoms form the inter-block chemical bonds
    // Dimsize: n_block_types x max_n_interblock_bonds
    TView<Int, 2, D> block_type_atoms_forming_chemical_bonds,

    // what is the path distance between pairs of atoms in the block
    // Dimsize: n_block_types x max_n_atoms x max_n_atoms
    TView<Int, 3, D> block_type_path_distance,
    //////////////////////

    // LJ parameters
    TView<LJTypeParams<Real>, 1, D> type_params,
    TView<LJGlobalParams<Real>, 1, D> global_params,
    TView<Real, 1, D> lj_lk_weights,
    TView<Real, 1, D> output,
    TView<int64_t, 1, tmol::Device::CPU> annealer) -> void {
  using tmol::pack::sim_anneal::compiled::RPECalc;
  using tmol::pack::sim_anneal::compiled::SimAnnealer;

  int64_t annealer_uint = annealer[0];
  SimAnnealer *sim_annealer = reinterpret_cast<SimAnnealer *>(annealer_uint);
  std::shared_ptr<RPECalc> calc =
      std::make_shared<LJRPECudaCalc<DeviceDispatch, D, Real, Int>>(
          context_coords,
          context_block_type,
          alternate_coords,
          alternate_ids,
          context_system_ids,
          system_min_bond_separation,
          system_inter_block_bondsep,
          system_neighbor_list,
          block_type_n_atoms,
          block_type_atom_types,
          block_type_n_interblock_bonds,
          block_type_atoms_forming_chemical_bonds,
          block_type_path_distance,
          type_params,
          global_params,
          lj_lk_weights,
          output);

  sim_annealer->add_score_component(calc);
}

template struct LJRPEDispatch<ForallDispatch, tmol::Device::CUDA, float, int>;
template struct LJRPEDispatch<ForallDispatch, tmol::Device::CUDA, double, int>;
template struct LJRPERegistratorDispatch<
    ForallDispatch,
    tmol::Device::CUDA,
    float,
    int>;
template struct LJRPERegistratorDispatch<
    ForallDispatch,
    tmol::Device::CUDA,
    double,
    int>;

}  // namespace potentials
}  // namespace ljlk
}  // namespace score
}  // namespace tmol
