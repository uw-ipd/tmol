#include "hip/hip_runtime.h"

// The location of CUDAGeneratorImpl changed in torch 1.11
#if TORCH_VERSION_MAJOR == 1 && TORCH_VERSION_MINOR < 11
#include <ATen/CUDAGeneratorImpl.h>
#else
#include <ATen/cuda/CUDAGeneratorImpl.h>
#endif

#include <ATen/Context.h>
// #include <THC/THCTensorRandom.h>
#include <c10/core/DeviceType.h>
#include <c10/cuda/CUDAStream.h>
//#include <THC/THCGenerator.hpp>

#include <tmol/utility/tensor/TensorAccessor.h>
#include <tmol/utility/tensor/TensorPack.h>
#include <tmol/utility/gpu_error_check.hh>

#include <tmol/score/common/accumulate.hh>
#include <tmol/score/common/forall_dispatch.cuda.impl.cuh>

#include "simulated_annealing.hh"

#include <hip/hip_cooperative_groups.h>
#include <moderngpu/cta_reduce.hxx>
#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/transform.hxx>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

#include <ctime>

// TEMP // Stolen from torch, v1.0.0
// TEMP // Expose part of the torch library that otherwise is
// TEMP // not part of the API.
// TEMP THCGenerator* THCRandom_getGenerator(THCState* state);
// TEMP
// TEMP // Stolen from torch, v1.0.0;
// TEMP // unnecessary in the latest release, where this function
// TEMP // is built in to CUDAGenerator.
// TEMP // Modified slightly as the input Generator is unused.
// TEMP // increment should be at least the number of hiprand() random numbers
// used in TEMP // each thread. TEMP std::pair<uint64_t, uint64_t>
// next_philox_seed(uint64_t increment) { TEMP   // static bool seeded = false;
// TEMP   // if ( ! seeded ) {
// TEMP   //   std::cout << "Setting RNG seed" << std::endl;
// TEMP   //   THCRandom_manualSeed(at::globalContext().getTHCState(), 0);
// TEMP   //   seeded = true;
// TEMP   // }
// TEMP   auto gen_ = THCRandom_getGenerator(at::globalContext().getTHCState());
// TEMP   uint64_t offset = gen_->state.philox_seed_offset.fetch_add(increment);
// TEMP   return std::make_pair(gen_->state.initial_seed, offset);
// TEMP }

namespace tmol {
namespace pack {
namespace sim_anneal {
namespace compiled {

hipStream_t packer_stream(0);
int count_pick_passes(0);
int count_mc_passes(0);

/// @brief Return a uniformly-distributed integer in the range
/// between 0 and n-1.
/// Note that hiprand_uniform() returns a random number in the range
/// (0,1], unlike unlike rand() returns a random number in the range
/// [0,1). Take care with hiprand_uniform().
__device__ inline int curand_in_range(
    hiprandStatePhilox4_32_10_t* state, int n) {
  return int(hiprand_uniform(state) * n) % n;
}

template <
    template <tmol::Device>
    class Dispatch,
    tmol::Device D,
    typename Real,
    typename Int>
struct PickRotamers {
  static auto f(
      TView<Real, 3, D> context_coords,
      TView<Int, 2, D> context_coord_offsets,
      TView<Int, 2, D> context_block_type,
      TView<Int, 1, D> pose_id_for_context,
      TView<Int, 1, D> n_rots_for_pose,
      TView<Int, 1, D> rot_offset_for_pose,
      TView<Int, 1, D> block_type_ind_for_rot,
      TView<Int, 1, D> block_ind_for_rot,
      TView<Real, 2, D> rotamer_coords,
      TView<Int, 1, D> rotamer_coord_offsets,
      TView<Real, 2, D> alternate_coords,
      TView<Int, 1, D> alternate_coord_offsets,
      TView<Int, 2, D> alternate_block_id,
      TView<Int, 1, D> random_rots,
      TView<Int, 1, D> block_type_n_atoms,
      Int max_n_atoms_per_block,
      TView<int64_t, 1, tmol::Device::CPU> annealer_event) -> void {
    // Increment the cuda generator and capture the set for this execution
    std::pair<uint64_t, uint64_t> rng_engine_inputs;
    auto gen = at::check_generator<at::CUDAGeneratorImpl>(
        at::cuda::detail::getDefaultCUDAGenerator());
    {
      // aquire lock when using random generators
      std::lock_guard<std::mutex> lock(gen->mutex_);
      rng_engine_inputs = gen->philox_engine_inputs(1);
    }

    // Increment the seed (and capture the current seed) for the
    // cuda generator. The number of calls to hiprand must be known
    // by this statement -- there will be only a single call to hiprand
    //
    // auto philox_seed = next_philox_seed(1);

    int const n_contexts = context_coords.size(0);
    int const max_n_blocks = context_coord_offsets.size(1);
    // int const max_n_atoms = context_coords.size(2);
    int const n_poses = pose_id_for_context.size(0);
    int const n_rots = block_type_ind_for_rot.size(0);

    assert(context_coords.size(2) == 3);
    assert(context_coord_offsets.size(0) == n_contexts);

    assert(context_block_type.size(0) == n_contexts);
    assert(context_block_type.size(1) == max_n_blocks);
    assert(n_rots_for_pose.size(0) == n_poses);
    assert(rot_offset_for_pose.size(0) == n_poses);
    assert(block_ind_for_rot.size(0) == n_rots);
    assert(rotamer_coords.size(1) == 3);
    assert(rotamer_coord_offsets.size(0) == n_rots);

    assert(random_rots.size(0) == n_contexts);
    assert(alternate_coords.size(1) == 3);
    assert(alternate_coord_offsets.size(0) == 2 * n_contexts);
    assert(alternate_block_id.size(0) == 2 * n_contexts);
    assert(alternate_block_id.size(1) == 3);

    auto select_rotamer = [=] MGPU_DEVICE(int i) {
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(rng_engine_inputs.first, i, rng_engine_inputs.second, &state);

      Int i_pose = pose_id_for_context[i];
      Int i_n_rots = n_rots_for_pose[i_pose];

      if (i_n_rots == 0) {
        alternate_block_id[i * 2][0] = -1;
        alternate_block_id[i * 2][1] = -1;
        alternate_block_id[i * 2][2] = -1;
        alternate_block_id[i * 2 + 1][0] = -1;
        alternate_block_id[i * 2 + 1][1] = -1;
        alternate_block_id[i * 2 + 1][2] = -1;
        random_rots[i] = -1;
      } else {
        Int i_rot_local = curand_in_range(&state, i_n_rots);
        Int i_rot_global = i_rot_local + rot_offset_for_pose[i_pose];
        Int i_block = block_ind_for_rot[i_rot_global];
        random_rots[i] = i_rot_global;

        alternate_block_id[i * 2][0] = i;
        alternate_block_id[i * 2][1] = i_block;
        alternate_block_id[i * 2][2] = context_block_type[i][i_block];
        alternate_block_id[i * 2 + 1][0] = i;
        alternate_block_id[i * 2 + 1][1] = i_block;
        alternate_block_id[i * 2 + 1][2] = block_type_ind_for_rot[i_rot_global];
      }
    };

    // auto random_rots_cpu_tp = TPack<Int, 1,
    // tmol::Device::CPU>::zeros({n_contexts}); auto random_rots_cpu =
    // random_rots_cpu_tp.view; hipMemcpy(&random_rots_cpu[0], &random_rots[0],
    // sizeof(Int) * n_contexts, hipMemcpyDeviceToHost); for (int i = 0; i <
    // n_contexts; ++i) {
    //   std::cout << " " << random_rots_cpu[i];
    // }
    // std::cout << std::endl;

    auto copy_rotamer_coords = [=] EIGEN_DEVICE_FUNC(int i) {
      Int alt_id = i / max_n_atoms_per_block;
      Int i_context = alternate_block_id[alt_id][0];
      Int i_block = alternate_block_id[alt_id][1];
      Int i_block_type = alternate_block_id[alt_id][2];
      if (i_block == -1) {
        return;
      }

      // pretend we're responsible for this atom; treat this like
      // our thread index; it's not, but, it'll do
      Int const quasi_atom_ind = (i % max_n_atoms_per_block);
      Int const i_block_n_atoms = block_type_n_atoms[i_block_type];

      if (alt_id % 2 == 0) {
        // strided iteration
        int const alt_offset = alternate_coord_offsets[alt_id];
        int const context_offset = context_coord_offsets[i_context][i_block];
        for (int j = 0; j < 3; ++j) {
          int j_count = j * max_n_atoms_per_block + quasi_atom_ind;
          int atom_id = j_count / 3;
          int dim = j_count % 3;
          if (atom_id < i_block_n_atoms) {
            alternate_coords[alt_offset + atom_id][dim] =
                context_coords[i_context][context_offset + atom_id][dim];
          }
        }
      } else {
        int const i_rot = random_rots[i_context];
        int const alt_offset = alternate_coord_offsets[alt_id];
        int const rotamer_offset = rotamer_coord_offsets[i_rot];
        // strided iteration
        for (int j = 0; j < 3; ++j) {
          int j_count = j * max_n_atoms_per_block + quasi_atom_ind;
          int atom_id = j_count / 3;
          int dim = j_count % 3;
          if (atom_id < i_block_n_atoms) {
            alternate_coords[alt_offset + atom_id][dim] =
                rotamer_coords[rotamer_offset + atom_id][dim];
          }
        }
      }
    };

    ++count_pick_passes;

    if (packer_stream == 0) {
      // packer_stream = at::cuda::getStreamFromPool().stream();
      hipStreamCreate(&packer_stream);
    }
    // mgpu::standard_context_t context(packer_stream);
    mgpu::standard_context_t context;
    // Dispatch<D>::forall(n_contexts, select_rotamer);
    mgpu::transform(select_rotamer, n_contexts, context);

    // Dispatch<D>::forall(n_contexts * 2 * max_n_atoms, copy_rotamer_coords);
    mgpu::transform(
        copy_rotamer_coords, n_contexts * 2 * max_n_atoms_per_block, context);

    // Record an event for the completion of the initialization of new
    // coordinates into the alternate_coords and alternate_block_id tensors so
    // that the score terms can wait until the rotamer coordinates are ready to
    // be evaluated
    if (annealer_event[0] != 0) {
      auto annealer_event_ptr =
          reinterpret_cast<hipEvent_t>(annealer_event[0]);
      hipEventRecord(annealer_event_ptr, context.stream());
      // std::cout << "Pick Rots " << count_pick_passes << ": recorded new event
      // " << annealer_event_ptr << " in stream " << context.stream() <<
      // std::endl;
    }
  }
};

void wait_on_score_events(
    hipStream_t stream, TView<int64_t, 1, tmol::Device::CPU> score_events) {
  int const n_score_terms = score_events.size(0);
  for (int i = 0; i < n_score_terms; ++i) {
    hipEvent_t event = reinterpret_cast<hipEvent_t>(score_events[i]);
    if (!event) {
      // not all entries in the score_events_ tensor are
      // non-null
      continue;
    }
    hipError_t status = hipEventQuery(event);
    // std::cout << "MC " << count_mc_passes << " Event " << event << " status "
    // << status << " (success =" << hipSuccess << ", ErrorNotReady=" <<
    // hipErrorNotReady << ")" << std::endl;
    if (status == hipSuccess) {
      // no need to wait
    } else if (status == hipErrorNotReady) {
      // std::cout << "MC AcceptReject " << count_mc_passes << " waiting on
      // event " << event << " in stream " << stream << std::endl;
      hipStreamWaitEvent(stream, event, 0);
    } else {
      // potential error situation?
    }
  }
}

template <
    template <tmol::Device>
    class Dispatch,
    tmol::Device D,
    typename Real,
    typename Int>
struct MetropolisAcceptReject {
  static auto f(
      TView<Real, 1, tmol::Device::CPU> temperature,
      TView<Real, 3, D> context_coords,
      TView<Int, 2, D> context_coord_offsets,
      TView<Int, 2, D> context_block_type,
      TView<Real, 2, D> alternate_coords,
      TView<Int, 1, D> alternate_coord_offsets,
      TView<Int, 2, D> alternate_ids,
      TView<Real, 2, D> rotamer_component_energies,
      TView<Int, 1, D> accept,
      TView<Int, 1, D> block_type_n_atoms,
      Int max_n_atoms_per_block,
      TView<int64_t, 1, tmol::Device::CPU> score_events) -> void {
    int const n_contexts = context_coords.size(0);
    int const n_terms = rotamer_component_energies.size(0);
    // int const max_n_atoms = context_coords.size(2);

    assert(context_coord_offsets.size(0) == n_contexts);
    assert(rotamer_component_energies.size(1) == 2 * n_contexts);
    // assert(alternate_coords.size(0) == 2 * n_contexts);
    // assert(alternate_coords.size(1) == max_n_atoms);
    assert(alternate_coords.size(1) == 3);
    assert(alternate_coord_offsets.size(0) == 2 * n_contexts);
    assert(alternate_ids.size(0) == 2 * n_contexts);
    assert(accept.size(0) == n_contexts);
    assert(score_events.size(0) == n_terms);

    // TEMP!!!
    // auto sum_energies_tp = TPack<Real, 1, D>::zeros({1});
    // auto sum_energies = sum_energies_tp.view;

    // auto accept_tp = TPack<Int, 1, D>::zeros({n_contexts});
    // auto accept = accept_tp.view;

    std::pair<uint64_t, uint64_t> rng_engine_inputs;
    auto gen = at::check_generator<at::CUDAGeneratorImpl>(
        at::cuda::detail::getDefaultCUDAGenerator());
    {
      // aquire lock when using random generators
      std::lock_guard<std::mutex> lock(gen->mutex_);
      rng_engine_inputs = gen->philox_engine_inputs(1);
    }
    // auto philox_seed = next_philox_seed(1);

    Real const temp = temperature[0];
    ++count_mc_passes;
    int const n_mc_passes = count_mc_passes;

    auto accept_reject = [=] MGPU_DEVICE(int i) {
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(rng_engine_inputs.first, i, rng_engine_inputs.second, &state);

      Real altE = 0;
      Real currE = 0;
      for (int j = 0; j < n_terms; ++j) {
        currE += rotamer_component_energies[j][2 * i];
        altE += rotamer_component_energies[j][2 * i + 1];
        rotamer_component_energies[j][2 * i] = 0;
        rotamer_component_energies[j][2 * i + 1] = 0;
      }
      // Real sumE = altE + currE;
      // score::common::accumulate<D, Real>::add(sum_energies[0], sumE);
      Real deltaE = altE - currE;
      Real rand_unif = hiprand_uniform(&state);
      Real prob_accept = temp > 0 ? exp(-1 * deltaE / temp) : 0;
      bool i_accept = deltaE < 0 || rand_unif < prob_accept;
      // if (n_mc_passes % 1000 == 1) {
      //   printf(
      //       "accept reject temp=%f tid=%d dE=%f runif=%f proba=%f
      //       iaccept=%d\n", temp, i, deltaE, rand_unif, prob_accept,
      //       i_accept);
      // }
      accept[i] = i_accept;
      if (i_accept) {
        int block_id = alternate_ids[2 * i + 1][1];
        context_block_type[i][block_id] = alternate_ids[2 * i + 1][2];
      }
    };

    auto copy_accepted_coords = [=] MGPU_DEVICE(int i) {
      // if (i == 0) {
      //   printf("n total atoms calc'd: %f\n", sum_energies[0]);
      // }
      int context_id = i / max_n_atoms_per_block;
      Int quasi_atom_ind = i % max_n_atoms_per_block;
      Int accepted = accept[context_id];

      if (accepted) {
        int block_id = alternate_ids[2 * context_id + 1][1];
        int block_type = alternate_ids[2 * context_id + 1][2];
        int n_atoms = block_type_n_atoms[block_type];
        int const context_offset = context_coord_offsets[context_id][block_id];
        int const alternate_offset =
            alternate_coord_offsets[2 * context_id + 1];
        for (int j = 0; j < 3; ++j) {
          int j_count = j * max_n_atoms_per_block + quasi_atom_ind;
          int atom_id = j_count / 3;
          int dim = j_count % 3;

          if (atom_id < n_atoms) {
            context_coords[context_id][context_offset + atom_id][dim] =
                alternate_coords[alternate_offset + atom_id][dim];
          }
        }
      }
    };

    // First we ensure that scoring has completed,
    // then we make the accept/reject decision for each
    // trajectory. Finally, we copy the coordinates
    // from the trajectories that have accepted substitutions
    // into the context_coords tensor.
    if (packer_stream == 0) {
      packer_stream = c10::cuda::getStreamFromPool().stream();
    }

    // mgpu::standard_context_t context(packer_stream);
    mgpu::standard_context_t context;
    wait_on_score_events(context.stream(), score_events);
    mgpu::transform(accept_reject, n_contexts, context);
    gpuErrchk(hipPeekAtLastError());
    mgpu::transform(
        copy_accepted_coords, n_contexts * max_n_atoms_per_block, context);
    gpuErrchk(hipPeekAtLastError());
  }
};

template <
    template <tmol::Device>
    class Dispatch,
    tmol::Device D,
    typename Real,
    typename Int>
struct FinalOp {
  static auto f() -> void { hipDeviceSynchronize(); }
};

template struct PickRotamers<
    score::common::ForallDispatch,
    tmol::Device::CUDA,
    float,
    int32_t>;
template struct PickRotamers<
    score::common::ForallDispatch,
    tmol::Device::CUDA,
    double,
    int32_t>;

template struct MetropolisAcceptReject<
    score::common::ForallDispatch,
    tmol::Device::CUDA,
    float,
    int32_t>;
template struct MetropolisAcceptReject<
    score::common::ForallDispatch,
    tmol::Device::CUDA,
    double,
    int32_t>;

template struct FinalOp<
    score::common::ForallDispatch,
    tmol::Device::CUDA,
    float,
    int32_t>;
template struct FinalOp<
    score::common::ForallDispatch,
    tmol::Device::CUDA,
    double,
    int32_t>;

}  // namespace compiled
}  // namespace sim_anneal
}  // namespace pack
}  // namespace tmol
