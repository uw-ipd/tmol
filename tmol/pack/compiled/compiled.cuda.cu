#include "hip/hip_runtime.h"
#include <c10/DeviceType.h>
#include <ATen/Context.h>
#include <ATen/CUDAGenerator.h>
#include <THC/THCGenerator.hpp>
#include <THC/THCTensorRandom.h>

#include <tmol/utility/tensor/TensorAccessor.h>
#include <tmol/utility/tensor/TensorPack.h>

// ??? #include "annealer.hh"
#include "simulated_annealing.hh"

#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/transform.hxx>
#include <hip/hip_cooperative_groups.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

#include <ctime>


// Stolen from torch, v1.0.0
// Expose part of the torch library that otherwise is
// not part of the API.
THCGenerator* THCRandom_getGenerator(THCState* state);

// Stolen from torch, v1.0.0;
// unnecessary in the latest release, where this function
// is built in to CUDAGenerator.
// Modified slightly as the input Generator is unused.
// increment should be at least the number of hiprand() random numbers used in
// each thread.
std::pair<uint64_t, uint64_t> next_philox_seed(uint64_t increment) {
  auto gen_ = THCRandom_getGenerator(at::globalContext().getTHCState());
  uint64_t offset = gen_->state.philox_seed_offset.fetch_add(increment);
  return std::make_pair(gen_->state.initial_seed, offset);
}


namespace tmol {
namespace pack {
namespace compiled {


/// @brief Return a uniformly-distributed integer in the range
/// between 0 and n-1.
/// Note that hiprand_uniform() returns a random number in the range
/// (0,1], unlike unlike rand() returns a random number in the range
/// [0,1). Take care with hiprand_uniform().
__device__
inline
int
curand_in_range(
  hiprandStatePhilox4_32_10_t * state,
  int n
)
{
  return int(hiprand_uniform(state)*n) % n;
}

template <unsigned int nthreads, typename T, typename F>
__device__
__inline__
T
reduce_shfl(
  cooperative_groups::thread_block_tile<nthreads> g,
  T val,
  F f
)
{
  for (unsigned int i = nthreads / 2; i > 0; i /= 2) {
    T const shfl_val = g.shfl_down(val, i);
    val = f(val, shfl_val);
  }
  // thread 0 shares its sum with everyone
  // so that there is no disagreement on the
  // partition function value
  val = g.shfl(val, 0);
  return val;
}

template <unsigned int nthreads, typename T, typename F>
__device__
__inline__
T
exclusive_scan_shfl(
  cooperative_groups::thread_block_tile<nthreads> g,
  T val,
  F f
)
{
  for (unsigned int i = 1; i <= nthreads; i *= 2) {
    T const shfl_val = g.shfl_up(val, i);
    if (i < g.thread_rank()) {
      val = f(shfl_val, val);
    }
  }
  val = g.shfl_up(val, 1);
  if (g.thread_rank() == 0) {
    val = 0;
  }
  return val;
}

template <unsigned int nthreads, typename T, typename F>
__device__
__inline__
T
inclusive_scan_shfl(
  cooperative_groups::thread_block_tile<nthreads> g,
  T val,
  F f
)
{
  for (unsigned int i = 1; i <= nthreads; i *= 2) {
    T const shfl_val = g.shfl_up(val, i);
    if (g.thread_rank() >= i) {
      val = f(shfl_val, val);
    }
  }
  return val;
}


template<tmol::Device D>
inline
__device__
void
set_quench_order(
  TView<int, 2, D> quench_order,
  int dim1_ind,
  hiprandStatePhilox4_32_10_t * state
){
  // Create a random permutation of all the rotamers
  // and visit them in this order to ensure all of them
  // are seen during the quench step
  int const nrots = quench_order.size(0);
  for (int i = 0; i < nrots; ++i) {
    quench_order[i][dim1_ind] = i;
  }
  for (int i = 0; i <= nrots-2; ++i) {
    int rand_offset = curand_in_range(state, nrots-i);
    int j = i + rand_offset;
    // swap i and j;
    int jval = quench_order[j][dim1_ind];
    quench_order[j][dim1_ind] = quench_order[i][dim1_ind];
    quench_order[i][dim1_ind] = jval;
  }
}


template<
  unsigned int nthreads,
  tmol::Device D,
  typename Real,
  typename Int
>
inline
#ifdef __HIPCC__
__device__
#endif
Real
total_energy_for_assignment_parallel(
  cooperative_groups::thread_block_tile<nthreads> g,
  TView<Int, 1, D> nrotamers_for_res,
  TView<Int, 1, D> oneb_offsets,
  TView<Int, 1, D> res_for_rot,
  TView<Int, 2, D> nenergies,
  TView<int64_t, 2, D> twob_offsets,
  TView<Real, 1, D> energy1b,
  TView<Real, 1, D> energy2b,
  TensorAccessor<Int, 1, D> rotamer_assignment
)
{
  Real totalE = 0;
  int const nres = nrotamers_for_res.size(0);
  for (int i = g.thread_rank(); i < nres; i += nthreads) {
    int const irot_local = rotamer_assignment[i];
    int const irot_global = irot_local + oneb_offsets[i];

    totalE += energy1b[irot_global];
  }

  for (int i = g.thread_rank(); i < nres; i += nthreads) {
    int const irot_local = rotamer_assignment[i];

    for (int j = i+1; j < nres; ++j) {
      int const jrot_local = rotamer_assignment[j];
      if (nenergies[i][j] == 0) {
	continue;
      }
      float ij_energy = energy2b[
	twob_offsets[i][j]
	+ nrotamers_for_res[j] * irot_local
	+ jrot_local
      ];
      totalE += ij_energy;
    }
  }
  totalE = reduce_shfl(g, totalE, mgpu::plus_t<float>());
  return totalE;
}


template <tmol::Device D>
struct AnnealerDispatch
{
  static
  auto
  forward(
    TView<int, 1, D> nrotamers_for_res,
    TView<int, 1, D> oneb_offsets,
    TView<int, 1, D> res_for_rot,
    TView<int, 2, D> nenergies,
    TView<int64_t, 2, D> twob_offsets,
    TView<float, 1, D> energy1b,
    TView<float, 1, D> energy2b
  )
    -> std::tuple<
      TPack<float, 1, D>,
      TPack<int, 2, D> >
  {
    clock_t start = clock();

    int const nres = nrotamers_for_res.size(0);
    int const nrotamers = res_for_rot.size(0);

    int const n_blocks = 5000;
    int const n_simA_threads = 32 * n_blocks;
    int const n_outer_iterations = 5;
    int const n_inner_iterations = nrotamers / 8;
    float const high_temp = 10;
    float const low_temp = 0.2;

    int const n_spbr = 10;
    int const n_ibr = 10;
    int const n_simA_results_to_keep = 5000;
    int const n_simA_expansions_for_faster = 1;
    int const n_faster_traj = n_simA_results_to_keep * n_simA_expansions_for_faster;
    int const n_faster_threads = 32 * n_faster_traj;
    int const faster_history_size = 4;

    auto scores_t = TPack<float, 1, D>::zeros({n_blocks});
    auto rotamer_assignments_t = TPack<int, 2, D>::zeros({n_blocks, nres});
    auto best_rotamer_assignments_t = TPack<int, 2, D>::zeros({n_blocks, nres});
    auto sorted_assignment_inds_t = TPack<int, 1, D>::zeros({n_blocks});
    auto quench_order_t = TPack<int, 2, D>::zeros({nrotamers, n_blocks});
    auto sorted_scores_t = TPack<float, 1, D>::zeros({n_blocks});
    auto faster_rotamer_assignments_t = TPack<int, 2, D>::zeros({n_faster_traj, nres});
    auto best_faster_rotamer_assignments_t = TPack<int, 2, D>::zeros({n_faster_traj, nres});
    auto faster_perturbed_assignments_t = TPack<int, 2, D>::zeros({n_faster_traj, nres});
    auto faster_assignment_history_t = TPack<int, 3, D>::zeros({n_faster_traj, faster_history_size, nres});

    // auto curr_pair_energies_t = TPack<float, 3, D>::zeros({nres, nres, n_simA_threads});
    // auto alt_energies_t = TPack<float, 2, D>::zeros({nres, n_simA_threads});

    auto scores = scores_t.view;
    auto rotamer_assignments = rotamer_assignments_t.view;
    auto best_rotamer_assignments = rotamer_assignments_t.view;
    auto sorted_assignment_inds = sorted_assignment_inds_t.view;
    auto quench_order = quench_order_t.view;
    auto sorted_scores = sorted_scores_t.view;
    auto faster_rotamer_assignments = faster_rotamer_assignments_t.view;
    auto best_faster_rotamer_assignments = best_faster_rotamer_assignments_t.view;
    auto faster_perturbed_assignments = faster_perturbed_assignments_t.view;
    auto faster_assignment_history = faster_assignment_history_t.view;

    // auto curr_pair_energies = curr_pair_energies_t.view;
    // auto alt_energies = alt_energies_t.view;

    // This code will work for future versions of the torch/aten libraries, but not
    // this one.
    // // Increment the cuda generator
    // // I know I need to increment this, but I am unsure by how much!
    // std::pair<uint64_t, uint64_t> rng_engine_inputs;
    // at::CUDAGenerator * gen = at::cuda::detail::getDefaultCUDAGenerator();
    // {
    //   std::lock_guard<std::mutex> lock(gen->mutex_);
    //   rng_engine_inputs = gen->philox_engine_inputs(nrotamers * 400 + nres);
    // }

    // Increment the seed (and capture the current seed) for the
    // cuda generator. The number of calls to hiprand must be known
    // by this statement.
    // 1: nrotmaers*400 = 20 outer loop * nrotamers * 20 inner loop
    // calls to either hiprand_uniform or hiprand_uniform4 in either
    // the quench / non-quench cycles +
    // 2: nres = the initial seed state of the system is created by
    // picking a single random rotamer per residue.
    auto philox_seed = next_philox_seed( n_outer_iterations * n_inner_iterations + nres);

    auto run_simulated_annealing = [=] MGPU_DEVICE (int thread_id){
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(
        philox_seed.first,
        thread_id,
        philox_seed.second,
        &state);

      cooperative_groups::thread_block_tile<32> g = cooperative_groups::tiled_partition<32>(
        cooperative_groups::this_thread_block());
      int const warp_id = thread_id / 32;
      if (g.thread_rank() == 0) {
	sorted_assignment_inds[warp_id] = warp_id;
      }

      for (int i = g.thread_rank(); i < nres; i += 32) {
        int const i_nrots = nrotamers_for_res[i];
        int chosen = int(hiprand_uniform(&state) * i_nrots) % i_nrots;
        rotamer_assignments[warp_id][i] = chosen;
        best_rotamer_assignments[warp_id][i] = chosen;
      }

      float temperature = high_temp;
      float best_energy = total_energy_for_assignment_parallel(g,
	nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
	energy1b, energy2b, rotamer_assignments[warp_id]
      );
      float current_total_energy = best_energy;
      int ntrials = 0;
      for (int i = 0; i < n_outer_iterations; ++i) {

	// if (g.thread_rank() == 0) {
	//   printf("top of outer loop %d currentE %f bestE %f temp %f\n", i, current_total_energy, best_energy, temperature);
	// }
        bool quench = false;
	int i_n_inner_iterations = n_inner_iterations;
	// Disable quench
        // if (i == n_outer_iterations - 1) {
	//   i_n_inner_iterations = nrotamers;
        //   quench = true;
        //   temperature = 1e-20;
        //   for (int j = g.thread_rank(); j < nres; j += 32) {
        //     rotamer_assignments[warp_id][j] = best_rotamer_assignments[warp_id][j];
        //   }
        //   current_total_energy = total_energy_for_assignment_parallel(g,
	//     nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
	//     energy1b, energy2b, rotamer_assignments[warp_id]
        //   );
        // }

        for (int j = 0; j < i_n_inner_iterations; ++j) {
          int ran_rot;
          float accept_prob(0);
          if (quench) {
	    if (g.thread_rank() == 0) {
              if (j % nrotamers == 0) {
                set_quench_order(quench_order, warp_id, &state);
              }
              ran_rot = quench_order[j%nrotamers][warp_id];
	    }
	    ran_rot = g.shfl(ran_rot, 0);
	    accept_prob = .5;
          } else {
            if (g.thread_rank() == 0) {
              float4 four_rands = hiprand_uniform4(&state);
              ran_rot = int(four_rands.x * nrotamers) % nrotamers;
              accept_prob = four_rands.y;
            }
            ran_rot = g.shfl(ran_rot, 0);
            accept_prob = g.shfl(accept_prob, 0);
          }
          int const ran_res = res_for_rot[ran_rot];
          int const local_prev_rot = rotamer_assignments[warp_id][ran_res];
          int const ran_res_nrots = nrotamers_for_res[ran_res];
          int const ran_res_rotamer_offset = oneb_offsets[ran_res];

          bool prev_rot_in_range = false;
	  int thread_w_prev_rot = 0;
          { // scope
            int const local_ran_rot_orig = ran_rot - ran_res_rotamer_offset;
            int const local_prev_rot_wrapped = local_ran_rot_orig < local_prev_rot ?
              local_prev_rot :
              local_prev_rot + ran_res_nrots;
            prev_rot_in_range = local_ran_rot_orig + 32 > local_prev_rot_wrapped;
	    thread_w_prev_rot = prev_rot_in_range ?
	      local_prev_rot_wrapped - local_ran_rot_orig : 0;
          }
          int const local_ran_rot = prev_rot_in_range ? (
            (ran_rot - ran_res_rotamer_offset + g.thread_rank()) % ran_res_nrots) :
            (g.thread_rank() == 0 ?
              local_prev_rot :
              (ran_rot - ran_res_rotamer_offset  + g.thread_rank() - 1) % ran_res_nrots);
          ran_rot = local_ran_rot + ran_res_rotamer_offset;


          // If there are fewer rotamers on this residue than there are threads
          // active in the warp, do not wrap and consider a rotamer more than once
          bool const this_thread_active = ran_res_nrots > g.thread_rank();
	  bool const this_thread_last_active = ran_res_nrots == g.thread_rank() || g.thread_rank() == 32 - 1;

          float new_e = 9999;
	  if (this_thread_active) {
	    new_e = energy1b[ran_rot];
	  }

          // Temp: iterate across all residues instead of just the
          // neighbors of ran_rot_res
          if (this_thread_active) {
            for (int k=0; k < nres; ++k) {
              if (k == ran_res || nenergies[ran_res][k] == 0) {
                // alt_energies[k][warp_id] = 0;
                continue;
              }
              int const local_k_rot = rotamer_assignments[warp_id][k];

              int64_t const k_ran_offset = twob_offsets[k][ran_res];
              //int const kres_nrots = nrotamers_for_res[k];

              new_e += energy2b[k_ran_offset + ran_res_nrots * local_k_rot + local_ran_rot];
            }
	  }

          float const min_e = reduce_shfl(g, new_e, mgpu::minimum_t<float>());
	  // printf("thread %d min_e %f\n", thread_id, min_e);
          float myexp = expf( -1 * ( new_e - min_e ) / temperature );
	  // printf("thread %d myexp %f\n", thread_id, myexp);
          float const partition = reduce_shfl(g, myexp, mgpu::plus_t<float>());
	  // printf("thread %d partition %f\n", thread_id, partition);
          float const myprob = this_thread_active ? myexp / partition : 0;
	  // printf("thread %d myprob %f\n", thread_id, myprob);
          float scan_prob = inclusive_scan_shfl(g, myprob, mgpu::plus_t<float>());
	  // printf("thread %d prev rotamer %d new rotamer %d new_e %f active? %d temp %f\n", thread_id, local_prev_rot, local_ran_rot, new_e, this_thread_active, temperature);
 	  // printf("thread %d myexp %f part %f myprob %f scan_prob %f accept_prob %f\n", thread_id, myexp, partition, myprob, scan_prob, accept_prob);
	  if ( this_thread_last_active ) {
	    // due to numerical imprecision, it's entirely likely that the scan probability
	    // for the last active thread to be slightly more or slightly less than 1,
	    // and we want to ensure that there's a winner for each thread.
	    scan_prob = 1;
	  }
          int accept_rank = ( this_thread_active && accept_prob <= scan_prob);
	  // printf("thread %d accept_rank %d\n", thread_id, accept_rank);
	  accept_rank = inclusive_scan_shfl(g, accept_rank, mgpu::plus_t<int>());
	  // printf("thread %d accept_rank after scan %d\n", thread_id, accept_rank);

	  bool accept = accept_rank == 1 && this_thread_active;
	  // printf("thread %d accept %d\n", thread_id, accept);
	  int const accept_thread = reduce_shfl(g, accept ? g.thread_rank() : -1, mgpu::maximum_t<int>());
	  // if (g.thread_rank() == 0) {
	  //   printf("thread %d accept_thread %d\n", thread_id, accept_thread);
	  // }

	  float prev_e = g.shfl(new_e, thread_w_prev_rot);
	  // printf("thread %d prev_e %f\n", thread_id, prev_e);

	  bool new_best = false;
          if (accept) {
	    float deltaE = new_e - prev_e;
	    // printf("deltaE: %f (%f - %f)\n", deltaE, new_e, prev_e);
            rotamer_assignments[warp_id][ran_res] = local_ran_rot;
            current_total_energy = current_total_energy + deltaE;
            // for (int k=0; k < nres; ++k) {
            //   float k_energy = alt_energies[k][thread_id];
            //   curr_pair_energies[ran_res][k][thread_id] = k_energy;
            //   curr_pair_energies[k][ran_res][thread_id] = k_energy;
            // }
            if (current_total_energy < best_energy) {
	      new_best = true;
              best_energy = current_total_energy;
            }
	  }
	  current_total_energy = g.shfl(current_total_energy, accept_thread);
	  new_best = g.shfl(new_best, accept_thread);
	  if (new_best) {
	    for (int k=g.thread_rank(); k < nres; k += 32) {
	      best_rotamer_assignments[warp_id][k] = rotamer_assignments[warp_id][k];
	    }
	    best_energy = current_total_energy; // g.shfl(best_energy, accept_thread);
	  }

          ++ntrials;
          if (ntrials > 1000) {
            ntrials = 0;
            current_total_energy = total_energy_for_assignment_parallel(g,
              nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
	      energy1b, energy2b, rotamer_assignments[warp_id]);
	    // if (g.thread_rank() == 0) {
	    //   printf("refresh total energy currentE %f\n", current_total_energy);
	    // }
          }

        } // end inner loop

	// geometric cooling toward 0.3
	// std::cout << "temperature " << temperature << " energy " <<
	//  total_energy_for_assignment(nrotamers_for_res, oneb_offsets,
	//    res_for_rot, nenergies, twob_offsets, energy1b, energy2b, my_rotamer_assignment) << std::endl;
	temperature = 0.35 * (temperature - low_temp) + low_temp;

      } // end outer loop


      float totalE = total_energy_for_assignment_parallel(g,
	nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
	energy1b, energy2b, rotamer_assignments[warp_id]
      );
      if (g.thread_rank() == 0) {
	scores[warp_id] = totalE;
	// printf("pre-sort: warp %d score %f\n", warp_id, totalE);
      }
    };

    // typedef typename conditional_typedef_t<
    //     launch_arg_t,
    //     launch_box_t<
    //         arch_20_cta<128, 11, 8>,
    //         arch_35_cta<128, 7, 5>,
    //         arch_52_cta<128, 11, 8> > >::type_t launch_t;

    // auto reindex_rotamer_assignments = [=] MGPU_DEVICE (int thread_id) {
    //   cooperative_groups::thread_block_tile<32> g = cooperative_groups::tiled_partition<32>(
    //     cooperative_groups::this_thread_block());
    //   int const warp_id = thread_id / 32;
    //   int const input_assignment = sorted_assignment_inds[warp_id];
    //   for (int i = g.thread_rank(); i < nres; i += 32) {
    // 	sorted_rotamer_assignments[warp_id][i] =
    // 	  rotamer_assignments[input_assignment][i];
    //   }
    // };

    philox_seed = next_philox_seed(n_spbr);

    // auto faster_sPBR = [=] MGPU_DEVICE (int thread_id) {
    //   hiprandStatePhilox4_32_10_t state;
    //   hiprand_init(
    //     philox_seed.first,
    //     thread_id,
    //     philox_seed.second,
    //     &state);
    //
    //   cooperative_groups::thread_block_tile<32> g = cooperative_groups::tiled_partition<32>(
    //     cooperative_groups::this_thread_block());
    //   int warp_id = thread_id / 32;
    //
    //   // 1 start from one of the top results of the previous run
    //   int prev_run_sorted_id = warp_id / n_simA_expansions_for_faster;
    //   int prev_run_id = sorted_assignment_inds[prev_run_sorted_id];
    //   for (int i = g.thread_rank(); i < nres; i += 32) {
    // 	faster_rotamer_assignments[warp_id][i] =
    // 	  rotamer_assignments[prev_run_id][i];
    //   }
    //   float energy = total_energy_for_assignment_parallel(g,
    // 	nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
    // 	energy1b, energy2b, faster_rotamer_assignments[warp_id]);
    //
    //   // 2. iterate
    //   for (int spbr_iteration = 0; spbr_iteration < n_spbr; ++spbr_iteration) {
    // 	// 3. pick a rotamer
    // 	int ran_rot;
    // 	if (g.thread_rank() == 0) {
    // 	  float rand_num = hiprand_uniform(&state);
    // 	  ran_rot = int(rand_num * nrotamers) % nrotamers;
    // 	}
    // 	ran_rot = g.shfl(ran_rot, 0);
    // 	int const ran_res = res_for_rot[ran_rot];
    // 	int const ran_res_nrots = nrotamers_for_res[ran_res];
    // 	int const ran_rot_local = ran_rot - oneb_offsets[ran_res];
    //
    // 	// initialize the perturbed assignments array for this iteration.
    // 	// many of these will be overwritten, but for memory access efficiency
    // 	// copy everything over now.
    // 	for (int i = g.thread_rank(); i < nres; ++i) {
    // 	  int irot = i == ran_res ? ran_rot_local :
    // 	    faster_rotamer_assignments[warp_id][i];
    // 	  faster_perturbed_assignments[warp_id][i] = irot;
    // 	}
    //
    // 	// 4. relax the neighbors of this residue
    // 	for (int i = 0; i < nres; ++i) {
    // 	  // 4a. Find the lowest energy rotamer for residue i
    // 	  if (ran_res == i || nenergies[ran_res][i] == 0) {
    // 	    continue;
    // 	  }
    //
    // 	  int my_best_rot = 0;
    // 	  int my_best_rot_E = 1e38; // hack! max float
    // 	  int i_nrots = nrotamers_for_res[i];
    // 	  for (int j = g.thread_rank(); j < i_nrots; ++j) {
    // 	    int const j_global = j + oneb_offsets[i];
    // 	    float jE = energy1b[j_global];
    // 	    for (int k = 0; k < nres; ++k) {
    // 	      if (k == i || nenergies[k][ran_rot] == 0) continue;
    //
    // 	      int const k_rotamer = k == ran_res ?
    // 		ran_rot : faster_rotamer_assignments[warp_id][k];
    // 	      jE += energy2b[
    // 		twob_offsets[k][ran_res] +
    // 		ran_res_nrots * k_rotamer + j];
    // 	    }
    //
    // 	    if (j == g.thread_rank() || jE < my_best_rot_E) {
    // 	      my_best_rot = j;
    // 	      my_best_rot_E = jE;
    // 	    }
    // 	  }
    // 	  // now all threads compare: who has the lowest energy
    // 	  float best_E = reduce_shfl(g, my_best_rot_E, mgpu::minimum_t<float>());
    // 	  int mine_is_best = best_E == my_best_rot_E;
    // 	  int scan_val = inclusive_scan_shfl(g, mine_is_best, mgpu::plus_t<int>());
    // 	  if (mine_is_best && scan_val == 1) {
    // 	    // exactly one thread saves the assigned rotamer to the
    // 	    // faster_perturbed_assignemnt array
    // 	    faster_perturbed_assignments[warp_id][i] = my_best_rot;
    // 	  }
    // 	}
    //
    // 	// 5. compute the new total energy after relaxation
    // 	float alt_energy = total_energy_for_assignment_parallel(g,
    // 	  nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
    // 	  energy1b, energy2b, faster_perturbed_assignments[warp_id]);
    //
    // 	// 6. if the energy decreases, accept the new
    // 	if (g.thread_rank() == 0) {
    // 	  printf("prevE %f newE %f\n", energy, alt_energy);
    // 	}
    // 	if (alt_energy < energy) {
    //
    // 	  energy = alt_energy;
    // 	  for (int i = g.thread_rank(); i < nres; i += 32) {
    // 	    faster_rotamer_assignments[warp_id][i] =
    // 	      faster_perturbed_assignments[warp_id][i];
    // 	  }
    // 	}
    //   }
    // };

    auto faster_iBR = [=] MGPU_DEVICE (int thread_id) {
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(
        philox_seed.first,
        thread_id,
        philox_seed.second,
        &state);

      cooperative_groups::thread_block_tile<32> g = cooperative_groups::tiled_partition<32>(
        cooperative_groups::this_thread_block());
      int warp_id = thread_id / 32;

      // 1 start from one of the top results of the previous run
      int prev_run_sorted_id = warp_id / n_simA_expansions_for_faster;
      int prev_run_id = sorted_assignment_inds[prev_run_sorted_id];
      for (int i = g.thread_rank(); i < nres; i += 32) {
	int i_rot = rotamer_assignments[prev_run_id][i];
	faster_rotamer_assignments[warp_id][i] = i_rot;
	best_faster_rotamer_assignments[warp_id][i] = i_rot;
      }
      float energy = total_energy_for_assignment_parallel(g,
	nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
	energy1b, energy2b, faster_rotamer_assignments[warp_id]);
      float best_energy = energy;

      // 2. iterate
      for (int ibr_iteration = 0; ibr_iteration < n_ibr; ++ibr_iteration) {

	bool converged = true;
	// 3. batch relax all residues
	for (int i = 0; i < nres; ++i) {
	  // 3a. Find the lowest energy rotamer for residue i
	  int i_nrots = nrotamers_for_res[i];
	  int i_curr_rot = faster_rotamer_assignments[warp_id][i];
	  int i_rot_offset = oneb_offsets[i];

	  int my_best_rot = 0;
	  float my_best_rot_E = 9999; // hack! max float
	  for (int j = g.thread_rank(); j < i_nrots; j += 32) {
	    int const j_global = j + i_rot_offset;
	    float jE = energy1b[j_global];
	    for (int k = 0; k < nres; ++k) {
	      if (k == i || nenergies[k][i] == 0) continue;

	      int const k_rotamer = faster_rotamer_assignments[warp_id][k];
	      jE += energy2b[
		twob_offsets[k][i] +
		i_nrots * k_rotamer + j];
	    }

	    if (j == g.thread_rank() || jE < my_best_rot_E) {
	      my_best_rot = j;
	      my_best_rot_E = jE;
	    }
	  }
	  // now all threads compare: who has the lowest energy
	  float best_rot_E = reduce_shfl(g, my_best_rot_E, mgpu::minimum_t<float>());
	  int mine_is_best = best_rot_E == my_best_rot_E;
	  int scan_val = inclusive_scan_shfl(g, mine_is_best, mgpu::plus_t<int>());
	  //printf("thread %d res %d curr rot %d my best rot %d E %e minebest %d scan %d\n",
	  //  g.thread_rank(), i, i_curr_rot, my_best_rot, my_best_rot_E, mine_is_best, scan_val);
	  if (mine_is_best && scan_val == 1) {
	    // exactly one thread saves the assigned rotamer to the
	    // faster_perturbed_assignemnt array
	    faster_perturbed_assignments[warp_id][i] = my_best_rot;
	    if (i_curr_rot != my_best_rot) {
	      converged = false;
	    }
	  }
	}

	// 4. compute the new total energy after relaxation
	float alt_energy = total_energy_for_assignment_parallel(g,
	  nrotamers_for_res, oneb_offsets, res_for_rot, nenergies, twob_offsets,
	  energy1b, energy2b, faster_perturbed_assignments[warp_id]);

	// 5. accept the new state regardless of energies; store the 
	// if (g.thread_rank() == 0) {
	//   printf("prevE %f newE %f\n", energy, alt_energy);
	// }
	energy = alt_energy;
	bool accept_as_best = energy < best_energy;
	if (accept_as_best) {
	  best_energy = energy;
	}
	for (int i = g.thread_rank(); i < nres; i += 32) {
	  int i_rot = faster_perturbed_assignments[warp_id][i];
	  faster_rotamer_assignments[warp_id][i] = i_rot;
	  if (accept_as_best) {
	    best_faster_rotamer_assignments[warp_id][i] = i_rot;
	  }
	}

	// quit if we have converged
	int all_converged = reduce_shfl(g, (int) converged, mgpu::minimum_t<int>());
	if (all_converged) {
	  break;
	}

	// Check this state against the history of states
	int const max_hist_to_check = min(faster_history_size, ibr_iteration-1);
	bool exit_ibr_loop = true;
	for (int i = 0; i < max_hist_to_check; ++i) {
	  int const i_hist = (ibr_iteration - i - 1) % faster_history_size;
	  exit_ibr_loop = true;
	  for (int j = g.thread_rank(); j < nres; j += 32) {
	    if (faster_rotamer_assignments[warp_id][j] !=
	      faster_assignment_history[warp_id][i_hist][j]) {
	      exit_ibr_loop = false;
	    }
	    // quit early if we found a mismatch, but only if
	    // all threads are currently active
	    if (j - g.thread_rank() + 32 < nres) {
	      exit_ibr_loop = reduce_shfl(g, exit_ibr_loop, mgpu::minimum_t<bool>());
	      if (! exit_ibr_loop) {
		break;
	      }
	    }
	  }
	  // we found a state in the history that matches exactly
	  // the newly adopted state
	  if (exit_ibr_loop) {
	    break;
	  }
	}
	// we have repeated a state assignment; ibr will walk us in circles from
	// here forward
	if (exit_ibr_loop) {
	  break;
	}

	// now save this rotamer assignment as part of the history
	// (overwriting some of that history)
	int const ibr_it_hist = ibr_iteration % faster_history_size;
	for (int i = g.thread_rank(); i < nres; i += 32) {
	  faster_assignment_history[warp_id][ibr_it_hist][i] =
	    faster_rotamer_assignments[warp_id][i];
	}

      } // end for ibr_iterations
      if (g.thread_rank() == 0) {
	scores[warp_id] = best_energy;
      }
    };


    mgpu::standard_context_t context;
    mgpu::transform<32, 1>(run_simulated_annealing, n_simA_threads, context);
    // mgpu::mergesort(
    //   scores.data(), sorted_assignment_inds.data(), n_blocks, mgpu::less_t<float>(), context);
    // mgpu::transform<128, 4>(reindex_rotamer_assignments, n_simA_threads, context);
    //mgpu::transform<32, 1>(faster_sPBR, n_faster_threads, context);
    mgpu::transform<32, 1>(faster_iBR, n_faster_threads, context);

    hipDeviceSynchronize();
    clock_t stop = clock();
    std::cout << "GPU simulated annealing in " <<
      ((double) stop - start)/CLOCKS_PER_SEC << std::endl;

    return {scores_t, best_faster_rotamer_assignments_t};
  }

};

template struct AnnealerDispatch<tmol::Device::CUDA>;

} // namespace compiled
} // namespace pack
} // namespace tmol
